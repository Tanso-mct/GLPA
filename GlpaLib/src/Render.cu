#include "hip/hip_runtime.h"
#include "Render.cuh"

#include "GlpaLog.h"
#include "GlpaConsole.h"

Glpa::Render2d::Render2d()
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "Constructor");
}

Glpa::Render2d::~Render2d()
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "Destructor");
}

void Glpa::Render2d::setBackground(std::string color, DWORD& bg)
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, color.c_str());
    if (color == Glpa::COLOR_BLACK)
    {
        Glpa::Color instColor(0, 0, 0, 1);
        bg = instColor.GetDword();
    }
    else if (color == Glpa::COLOR_GREEN)
    {
        Glpa::Color instColor(0, 200, 0, 1);
        bg = instColor.GetDword();
    }
    else
    {
        Glpa::Color instColor(0, 200, 0, 1);
        bg = instColor.GetDword();
    }
}

void Glpa::Render2d::editObjsPos(Glpa::Image *img){
    if (!malloc) return;

    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, img->getName().c_str());
    hipFree(dImgPosX);
    hipFree(dImgPosY);

    int index = std::distance(imgNames.begin(), std::find(imgNames.begin(), imgNames.end(), img->getName()));

    Vec2d imgPos = img->GetPos();
    hImgPosX[index] = imgPos.x;
    hImgPosY[index] = imgPos.y;

    hipMalloc(&dImgPosX, hImgPosX.size() * sizeof(int));
    hipMemcpy(dImgPosX, hImgPosX.data(), hImgPosX.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgPosY, hImgPosY.size() * sizeof(int));
    hipMemcpy(dImgPosY, hImgPosY.data(), hImgPosY.size() * sizeof(int), hipMemcpyHostToDevice);

}

void Glpa::Render2d::editBufSize(int bufWidth, int bufHeight, int bufDpi)
{
    if (!malloc) return;

    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, "");
    hipFree(dBuf);
    hipMalloc(&dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD));
}

void Glpa::Render2d::dMalloc
(
    std::unordered_map<std::string, Glpa::SceneObject*>& objs,
    std::map<int, std::vector<std::string>>& drawOrderMap, std::vector<std::string>& drawOrder,
    int bufWidth, int bufHeight, int bufDpi, std::string bgColor
){
    if (malloced) return;

    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, "");

    maxImgWidth = 0;
    maxImgHeight = 0;
    
    hImgPosX.clear();
    hImgPosY.clear();
    hImgWidth.clear();
    hImgHeight.clear();

    imgNames.clear();
    drawOrder.clear();

    hImgData.clear();
    for (int i = 0; i < hImgData.size(); i++)
    {
        hipFree(hImgData[i]);
    }

    for (auto& pair : drawOrderMap)
    {
        for (int i = 0; i < pair.second.size(); i++)
        {
            if (Glpa::Image* img = dynamic_cast<Glpa::Image*>(objs[pair.second[i]]))
            {
                if (img->getVisible())
                {
                    Vec2d imgPos = img->GetPos();
                    hImgPosX.push_back(imgPos.x);
                    hImgPosY.push_back(imgPos.y);
                    hImgWidth.push_back(img->GetWidth());
                    hImgHeight.push_back(img->GetHeight());

                    imgNames.push_back(img->getName());
                    drawOrder.push_back(img->getName());

                    maxImgWidth = (maxImgWidth < img->GetWidth()) ? img->GetWidth() : maxImgWidth;
                    maxImgHeight = (maxImgHeight < img->GetHeight()) ? img->GetHeight() : maxImgHeight;

                    LPDWORD dData;
                    int dataSize = img->GetWidth() * img->GetHeight() * sizeof(DWORD);
                    hipMalloc(&dData, dataSize);
                    hipMemcpy(dData, img->GetData(), dataSize, hipMemcpyHostToDevice);
                    hImgData.push_back(dData);
                }
            }
        }
    }

    imgAmount = hImgData.size();

    setBackground(bgColor, backgroundColor);

    hipMalloc(&dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD));

    hipMalloc(&dImgPosX, hImgPosX.size() * sizeof(int));
    hipMemcpy(dImgPosX, hImgPosX.data(), hImgPosX.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgPosY, hImgPosY.size() * sizeof(int));
    hipMemcpy(dImgPosY, hImgPosY.data(), hImgPosY.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgWidth, hImgWidth.size() * sizeof(int));
    hipMemcpy(dImgWidth, hImgWidth.data(), hImgWidth.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgHeight, hImgHeight.size() * sizeof(int));
    hipMemcpy(dImgHeight, hImgHeight.data(), hImgHeight.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgData, hImgData.size() * sizeof(DWORD*));
    hipMemcpy(dImgData, hImgData.data(), hImgData.size() * sizeof(DWORD*), hipMemcpyHostToDevice);

    malloced = true;

}

void Glpa::Render2d::dRelease()
{
    if (!malloced) return;

    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, "");

    hipFree(dBuf);
    hipFree(dImgPosX);
    hipFree(dImgPosY);
    hipFree(dImgWidth);
    hipFree(dImgHeight);

    for (int i = 0; i < hImgData.size(); i++)
    {
        LPDWORD ptDeviceData;
    
        hipMemcpy(&ptDeviceData, &dImgData[i], sizeof(LPDWORD), hipMemcpyDeviceToHost);
        hipFree(ptDeviceData);
    }

    hipFree(dImgData);

    malloced = false;
}

__global__ void Gpu2dDraw
(
    int *imgPosX, int *imgPosY, int* imgWidth, int* imgHeight, LPDWORD *imgData, int imgAmount,
    LPDWORD buf, int bufWidth, int bufHeight, int bufDpi, DWORD background
){
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int y = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < bufWidth)
    {
        if (y < bufHeight)
        {
            atomicExch((unsigned int*)&buf[x + y * bufWidth * bufDpi], (unsigned int)background);

            int isBackgroundIF = TRUE;
            int isNotBackgroundIF = FALSE;
            for (int i = 0; i < imgAmount; i++)
            {
                int xInImgIF = (x >= imgPosX[i] && x < imgPosX[i] + imgWidth[i]) ? TRUE : FALSE;
                int yInImgIF = (y >= imgPosY[i] && y < imgPosY[i] + imgHeight[i]) ? TRUE : FALSE;

                int writeIF = (xInImgIF == TRUE && yInImgIF == TRUE) ? TRUE : FALSE;

                for (int cb1 = 0; cb1 < writeIF; cb1++)
                {
                    int imgX = x - imgPosX[i];
                    int imgY = y - imgPosY[i];

                    for (int cb2 = 0; cb2 < isNotBackgroundIF; cb2++)
                    {
                        BYTE bufR = (buf[x + y * bufWidth * bufDpi] >> 16) & 0xFF;
                        BYTE bufG = (buf[x + y * bufWidth * bufDpi] >> 8) & 0xFF;
                        BYTE bufB = buf[x + y * bufWidth * bufDpi] & 0xFF;

                        BYTE imgA = (imgData[i][imgX + imgY * imgWidth[i]] >> 24) & 0xFF;
                        BYTE imgR = (imgData[i][imgX + imgY * imgWidth[i]] >> 16) & 0xFF;
                        BYTE imgG = (imgData[i][imgX + imgY * imgWidth[i]] >> 8) & 0xFF;
                        BYTE imgB = imgData[i][imgX + imgY * imgWidth[i]] & 0xFF;

                        float alpha = static_cast<float>(imgA) / 255.0f;
                        float invAlpha = 1.0f - alpha;

                        bufR = static_cast<unsigned char>(alpha * imgR + invAlpha * bufR);
                        bufG = static_cast<unsigned char>(alpha * imgG + invAlpha * bufG);
                        bufB = static_cast<unsigned char>(alpha * imgB + invAlpha * bufB);

                        DWORD newColor = (1 << 24) | (bufR << 16) | (bufG << 8) | bufB;
                        atomicExch((unsigned int*)&buf[x + y * bufWidth * bufDpi], (unsigned int)newColor);
                    }

                    for (int cb2 = 0; cb2 < isBackgroundIF; cb2++)
                    {
                        atomicExch((unsigned int*)&buf[x + y * bufWidth * bufDpi], (unsigned int)imgData[i][imgX + imgY * imgWidth[i]]);
                        isBackgroundIF = FALSE;
                        isNotBackgroundIF = TRUE;
                    }

                }
            }
        }
    }
}

void Glpa::Render2d::run
(
        std::unordered_map<std::string, Glpa::SceneObject*>& objs,
        std::map<int, std::vector<std::string>>& drawOrderMap, std::vector<std::string>& drawOrder,
        LPDWORD buf, int bufWidth, int bufHeight, int bufDpi, std::string bgColor
){
    dMalloc(objs, drawOrderMap, drawOrder, bufWidth, bufHeight, bufDpi, bgColor);

    if (imgAmount != 0)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int dataSizeY = bufWidth;
        int dataSizeX = bufHeight;

        int desiredThreadsPerBlockX = 16;
        int desiredThreadsPerBlockY = 16;

        int blocksX = (dataSizeX + desiredThreadsPerBlockX - 1) / desiredThreadsPerBlockX;
        int blocksY = (dataSizeY + desiredThreadsPerBlockY - 1) / desiredThreadsPerBlockY;

        blocksX = min(blocksX, deviceProp.maxGridSize[0]);
        blocksY = min(blocksY, deviceProp.maxGridSize[1]);

        int threadsPerBlockX = min(desiredThreadsPerBlockX, deviceProp.maxThreadsDim[0]);
        int threadsPerBlockY = min(desiredThreadsPerBlockY, deviceProp.maxThreadsDim[1]);

        dim3 dimBlock(threadsPerBlockX, threadsPerBlockY);
        dim3 dimGrid(blocksX, blocksY);

        Gpu2dDraw<<<dimGrid, dimBlock>>>
        (
            dImgPosX, dImgPosY, dImgWidth, dImgHeight, dImgData, imgAmount, 
            dBuf, bufWidth, bufHeight, bufDpi, backgroundColor
        );
        hipError_t error = hipDeviceSynchronize();
        if (error != 0){
            Glpa::runTimeError(__FILE__, __LINE__, {"Processing with Cuda failed."});
        }

        hipMemcpy(buf, dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD), hipMemcpyDeviceToHost);
    }
    else
    {
        hipMemcpy(buf, dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD), hipMemcpyDeviceToHost);
    }
}

Glpa::Render3d::Render3d()
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "Constructor");
}

Glpa::Render3d::~Render3d()
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "Destructor");
}

void Glpa::Render3d::dMallocCam(Glpa::Camera& cam)
{
    if (camMalloced) dReleaseCam();
    hipError_t err;

    err = hipMalloc(&dCamData, sizeof(Glpa::GPU_CAMERA));
    err = hipMemcpy(dCamData, &cam.getData(), sizeof(Glpa::GPU_CAMERA), hipMemcpyHostToDevice);

    camMalloced = true;
}

void Glpa::Render3d::dReleaseCam()
{
    if (!camMalloced) return;
    hipError_t err;

    err = hipFree(dCamData);

    camMalloced = false;
}

void Glpa::Render3d::dMallocObjsMtData
(
    std::unordered_map<std::string, Glpa::SceneObject*>& objs,
    std::unordered_map<std::string, Glpa::Material*>& mts
){
    if (objMtDataMalloced) return;
    hipError_t err;
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "");

    // Material data
    std::vector<Glpa::GPU_MATERIAL> hMts;
    int mtId = 0;
    for (auto& pair : mts)
    {
        Glpa::GPU_MATERIAL mt;
        err = hipMalloc
        (
            &mt.baseColor, 
            pair.second->GetMtWidth(Glpa::MATERIAL_DIFFUSE) * pair.second->GetMtHeight(Glpa::MATERIAL_DIFFUSE) * sizeof(DWORD)
        );
        err = hipMemcpy
        (
            mt.baseColor, 
            pair.second->GetMtData(Glpa::MATERIAL_DIFFUSE), 
            pair.second->GetMtWidth(Glpa::MATERIAL_DIFFUSE) * pair.second->GetMtHeight(Glpa::MATERIAL_DIFFUSE) * sizeof(DWORD), 
            hipMemcpyHostToDevice
        );

        mtIdMap[pair.first] = mtId;

        hMts.push_back(mt);
        mtId++;
    }
    err = hipMalloc(&dMts, mtId * sizeof(Glpa::GPU_MATERIAL));
    err = hipMemcpy(dMts, hMts.data(), mtId * sizeof(Glpa::GPU_MATERIAL), hipMemcpyHostToDevice);

    for (int i = 0; i < hMts.size(); i++)
    {
        err = hipFree(hMts[i].baseColor);
    }

    // Object data
    std::vector<Glpa::GPU_OBJECT3D_DATA> hObjData;
    int objId = 0;
    for (auto& pair : objs)
    {
        if (Glpa::StationaryObject* obj = dynamic_cast<Glpa::StationaryObject*>(pair.second))
        {
            Glpa::GPU_OBJECT3D_DATA objData;
            objData.id = objId;
            objData.mtId = mtIdMap[obj->GetMaterial()->getName()];

            std::vector<Glpa::GPU_POLYGON> polygons = obj->getPolyData();
            err = hipMalloc(&objData.polygons, polygons.size() * sizeof(Glpa::GPU_POLYGON));
            err = hipMemcpy(objData.polygons, polygons.data(), polygons.size() * sizeof(Glpa::GPU_POLYGON), hipMemcpyHostToDevice);

            objData.range = obj->getRangeRectData();

            objIdMap[pair.first] = objId;

            hObjData.push_back(objData);
            objId++;
        }
    }

    err = hipMalloc(&dObjData, objId * sizeof(Glpa::GPU_OBJECT3D_DATA));
    err = hipMemcpy(dObjData, hObjData.data(), objId * sizeof(Glpa::GPU_OBJECT3D_DATA), hipMemcpyHostToDevice);

    for (int i = 0; i < hObjData.size(); i++)
    {
        err = hipFree(hObjData[i].polygons);
    }

    objMtDataMalloced = true;
}

void Glpa::Render3d::dReleaseObjsMtData()
{
    if (!objMtDataMalloced) return;
    hipError_t err;
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "");

    for (int i = 0; i < mtIdMap.size(); i++)
    {
        LPDWORD dBaseColor = nullptr;
        err = hipMemcpy(dBaseColor, &dMts[i].baseColor, sizeof(LPDWORD), hipMemcpyDeviceToHost);
        err = hipFree(dBaseColor);
    }
    err = hipFree(dMts);

    for (int i = 0; i < objIdMap.size(); i++)
    {
        Glpa::GPU_POLYGON* dPolygons = nullptr;
        err = hipMemcpy(dPolygons, &dObjData[i].polygons, sizeof(Glpa::GPU_OBJECT3D_DATA*), hipMemcpyDeviceToHost);
        err = hipFree(dPolygons);
    }
    err = hipFree(dObjData);
    objMtDataMalloced = false;
}

void Glpa::Render3d::dMallocObjInfo(std::unordered_map<std::string, Glpa::SceneObject *> &objs)
{
    hipError_t err;
    if (objInfoMalloced)
    {
        dReleaseObjInfo();
    };

    std::vector<Glpa::GPU_OBJECT3D_INFO> hObjInfo;
    for (auto& pair : objIdMap)
    {
        if (Glpa::StationaryObject* obj = dynamic_cast<Glpa::StationaryObject*>(objs[pair.first]))
        {
            hObjInfo.push_back(obj->getInfo());
        }
        else
        {
            Glpa::outputErrorLog(__FILE__, __LINE__, {"Object is not a StationaryObject."});
        }
    }

    err = hipMalloc(&dObjInfo, objIdMap.size() * sizeof(Glpa::GPU_OBJECT3D_INFO));
    err = hipMemcpy(dObjInfo, hObjInfo.data(), objIdMap.size() * sizeof(Glpa::GPU_OBJECT3D_INFO), hipMemcpyHostToDevice);

    objInfoMalloced = true;
}

void Glpa::Render3d::dReleaseObjInfo()
{
    hipError_t err;
    if (!objInfoMalloced) return;

    err = hipFree(dObjInfo);
    objInfoMalloced = false;
}

__global__ void GpuPrepareObj
(
    Glpa::GPU_OBJECT3D_DATA* objData,
    Glpa::GPU_OBJECT3D_INFO* objInfo,
    Glpa::GPU_CAMERA* camData,
    int objAmount  
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    Glpa::GPU_VECTOR_MG vecMg;

    if (i < objAmount)
    {
        // Get the object's existence range in the camera coordinate system
        Glpa::GPU_RANGE_RECT objRangeRect;
        for (int vI = 0; vI < 8; vI++)
        {
            objRangeRect.addRangeV(camData->mtTransRot.productLeft3x1(objData[i].range.wv[vI]));
        }

        // By looking two-dimensionally, 
        // it is possible to determine whether an object is even partially within the camera's viewing angle.
        Glpa::GPU_VEC_2D cullingVecs[4] = {
            {objRangeRect.origin.x, objRangeRect.opposite.z},
            {objRangeRect.opposite.x, objRangeRect.opposite.z},
            {objRangeRect.origin.y, objRangeRect.opposite.z},
            {objRangeRect.opposite.y, objRangeRect.opposite.z}
        };

        Glpa::GPU_VEC_2D axisVec(0, -1);

        float vecsCos[4] = {
            vecMg.cos(cullingVecs[0], axisVec),
            vecMg.cos(cullingVecs[1], axisVec),
            vecMg.cos(cullingVecs[2], axisVec),
            vecMg.cos(cullingVecs[3], axisVec)
        };

        GPU_BOOL isObjZIn = GPU_CO
        (
            objRangeRect.origin.z >= -camData->farZ && objRangeRect.opposite.z <= -camData->nearZ, 
            TRUE, FALSE
        );

        GPU_BOOL isObjXzIn = GPU_CO
        (
            (objRangeRect.origin.x >= 0 && vecsCos[0] >= camData->fovXzCos) || 
            (objRangeRect.opposite.x <= 0 && vecsCos[1] >= camData->fovXzCos) ||
            (objRangeRect.origin.x <= 0 && objRangeRect.opposite.x >= 0),
            TRUE, FALSE
        );

        GPU_BOOL isObjYzIn = GPU_CO
        (
            (objRangeRect.origin.y >= 0 && vecsCos[2] >= camData->fovYzCos) || 
            (objRangeRect.opposite.y <= 0 && vecsCos[3] >= camData->fovYzCos) ||
            (objRangeRect.origin.y <= 0 && objRangeRect.opposite.y >= 0),
            TRUE, FALSE
        );

        objInfo[i].isInVV = GPU_CO
        (
            isObjZIn == TRUE && isObjXzIn == TRUE && isObjYzIn == TRUE, 
            TRUE, FALSE
        );

    } // if (i < objAmount)
}

void Glpa::Render3d::prepareObjs()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int dataSize = objIdMap.size();
    int desiredThreadsPerBlock = 256;

    int blocks = (dataSize + desiredThreadsPerBlock - 1) / desiredThreadsPerBlock;
    int threadsPerBlock = std::min(desiredThreadsPerBlock, deviceProp.maxThreadsPerBlock);

    dim3 dimBlock(threadsPerBlock);
    dim3 dimGrid(blocks);

    GpuPrepareObj<<<dimGrid, dimBlock>>>(dObjData, dObjInfo, dCamData, dataSize);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != 0) Glpa::runTimeError(__FILE__, __LINE__, {"Processing with Cuda failed."});
}

__global__ void GpuSetVs
(
    Glpa::GPU_OBJECT3D_DATA* objData,
    Glpa::GPU_OBJECT3D_INFO* objInfo,
    Glpa::GPU_CAMERA* camData,
    int objAmount  
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    Glpa::GPU_VECTOR_MG vecMg;

    if (i < objAmount)
    {
        
    }


}

void Glpa::Render3d::setVs()
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int dataSize = objIdMap.size();
    int desiredThreadsPerBlock = 256;

    int blocks = (dataSize + desiredThreadsPerBlock - 1) / desiredThreadsPerBlock;
    int threadsPerBlock = std::min(desiredThreadsPerBlock, deviceProp.maxThreadsPerBlock);

    dim3 dimBlock(threadsPerBlock);
    dim3 dimGrid(blocks);

    GpuSetVs<<<dimGrid, dimBlock>>>(dObjData, dObjInfo, dCamData, dataSize);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != 0) Glpa::runTimeError(__FILE__, __LINE__, {"Processing with Cuda failed."});
}

void Glpa::Render3d::rasterize()
{
}

void Glpa::Render3d::run(
    std::unordered_map<std::string, Glpa::SceneObject *> &objs, std::unordered_map<std::string, Glpa::Material *> &mts,
    Glpa::Camera &cam, LPDWORD buf, int bufWidth, int bufHeight, int bufDpi)
{
    dMallocCam(cam);
    if (!objMtDataMalloced) dMallocObjsMtData(objs, mts);
    dMallocObjInfo(objs);

    prepareObjs();
    setVs();
    rasterize();
}

void Glpa::Render3d::dRelease()
{
    dReleaseCam();
    dReleaseObjsMtData();
    dReleaseObjInfo();
}
