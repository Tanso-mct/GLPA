#include "hip/hip_runtime.h"
#include "Render.cuh"
#include "GlpaLog.h"

Glpa::Render2d::Render2d()
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "Constructor");
}

Glpa::Render2d::~Render2d()
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "Destructor");
}

void Glpa::Render2d::setBackground(std::string color, DWORD& bg)
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, color.c_str());
    if (color == Glpa::COLOR_BLACK)
    {
        Glpa::Color instColor(0, 0, 0, 1);
        bg = instColor.GetDword();
    }
    else if (color == Glpa::COLOR_GREEN)
    {
        Glpa::Color instColor(0, 200, 0, 1);
        bg = instColor.GetDword();
    }
    else
    {
        Glpa::Color instColor(0, 200, 0, 1);
        bg = instColor.GetDword();
    }
}

void Glpa::Render2d::editObjsPos(Glpa::Image *img){
    if (!malloc) return;

    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, img->getName().c_str());
    hipFree(dImgPosX);
    hipFree(dImgPosY);

    int index = std::distance(imgNames.begin(), std::find(imgNames.begin(), imgNames.end(), img->getName()));

    Vec2d imgPos = img->GetPos();
    hImgPosX[index] = imgPos.x;
    hImgPosY[index] = imgPos.y;

    hipMalloc(&dImgPosX, hImgPosX.size() * sizeof(int));
    hipMemcpy(dImgPosX, hImgPosX.data(), hImgPosX.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgPosY, hImgPosY.size() * sizeof(int));
    hipMemcpy(dImgPosY, hImgPosY.data(), hImgPosY.size() * sizeof(int), hipMemcpyHostToDevice);

}

void Glpa::Render2d::editBufSize(int bufWidth, int bufHeight, int bufDpi)
{
    if (!malloc) return;

    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, "");
    hipFree(dBuf);
    hipMalloc(&dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD));
}

void Glpa::Render2d::dMalloc
(
    std::unordered_map<std::string, Glpa::SceneObject*>& objs,
    std::map<int, std::vector<std::string>>& drawOrderMap, std::vector<std::string>& drawOrder,
    int bufWidth, int bufHeight, int bufDpi, std::string bgColor
){
    if (malloced) return;

    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, "");

    maxImgWidth = 0;
    maxImgHeight = 0;
    
    hImgPosX.clear();
    hImgPosY.clear();
    hImgWidth.clear();
    hImgHeight.clear();

    imgNames.clear();
    drawOrder.clear();

    hImgData.clear();
    for (int i = 0; i < hImgData.size(); i++)
    {
        hipFree(hImgData[i]);
    }

    for (auto& pair : drawOrderMap)
    {
        for (int i = 0; i < pair.second.size(); i++)
        {
            if (Glpa::Image* img = dynamic_cast<Glpa::Image*>(objs[pair.second[i]]))
            {
                if (img->getVisible())
                {
                    Vec2d imgPos = img->GetPos();
                    hImgPosX.push_back(imgPos.x);
                    hImgPosY.push_back(imgPos.y);
                    hImgWidth.push_back(img->GetWidth());
                    hImgHeight.push_back(img->GetHeight());

                    imgNames.push_back(img->getName());
                    drawOrder.push_back(img->getName());

                    maxImgWidth = (maxImgWidth < img->GetWidth()) ? img->GetWidth() : maxImgWidth;
                    maxImgHeight = (maxImgHeight < img->GetHeight()) ? img->GetHeight() : maxImgHeight;

                    LPDWORD dData;
                    int dataSize = img->GetWidth() * img->GetHeight() * sizeof(DWORD);
                    hipMalloc(&dData, dataSize);
                    hipMemcpy(dData, img->GetData(), dataSize, hipMemcpyHostToDevice);
                    hImgData.push_back(dData);
                }
            }
        }
    }

    imgAmount = hImgData.size();

    setBackground(bgColor, backgroundColor);

    hipMalloc(&dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD));

    hipMalloc(&dImgPosX, hImgPosX.size() * sizeof(int));
    hipMemcpy(dImgPosX, hImgPosX.data(), hImgPosX.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgPosY, hImgPosY.size() * sizeof(int));
    hipMemcpy(dImgPosY, hImgPosY.data(), hImgPosY.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgWidth, hImgWidth.size() * sizeof(int));
    hipMemcpy(dImgWidth, hImgWidth.data(), hImgWidth.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgHeight, hImgHeight.size() * sizeof(int));
    hipMemcpy(dImgHeight, hImgHeight.data(), hImgHeight.size() * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&dImgData, hImgData.size() * sizeof(DWORD*));
    hipMemcpy(dImgData, hImgData.data(), hImgData.size() * sizeof(DWORD*), hipMemcpyHostToDevice);

    malloced = true;

}

void Glpa::Render2d::dRelease()
{
    if (!malloced) return;

    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_RENDER, "");

    hipFree(dBuf);
    hipFree(dImgPosX);
    hipFree(dImgPosY);
    hipFree(dImgWidth);
    hipFree(dImgHeight);

    for (int i = 0; i < hImgData.size(); i++)
    {
        LPDWORD ptDeviceData;
    
        hipMemcpy(&ptDeviceData, &dImgData[i], sizeof(LPDWORD), hipMemcpyDeviceToHost);
        hipFree(ptDeviceData);
    }

    hipFree(dImgData);

    malloced = false;
}

void Glpa::Render2d::run
(
        std::unordered_map<std::string, Glpa::SceneObject*>& objs,
        std::map<int, std::vector<std::string>>& drawOrderMap, std::vector<std::string>& drawOrder,
        LPDWORD buf, int bufWidth, int bufHeight, int bufDpi, std::string bgColor
){
    dMalloc(objs, drawOrderMap, drawOrder, bufWidth, bufHeight, bufDpi, bgColor);

    if (imgAmount != 0)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int dataSizeY = bufWidth;
        int dataSizeX = bufHeight;

        int desiredThreadsPerBlockX = 16;
        int desiredThreadsPerBlockY = 16;

        int blocksX = (dataSizeX + desiredThreadsPerBlockX - 1) / desiredThreadsPerBlockX;
        int blocksY = (dataSizeY + desiredThreadsPerBlockY - 1) / desiredThreadsPerBlockY;

        blocksX = min(blocksX, deviceProp.maxGridSize[0]);
        blocksY = min(blocksY, deviceProp.maxGridSize[1]);

        int threadsPerBlockX = min(desiredThreadsPerBlockX, deviceProp.maxThreadsDim[0]);
        int threadsPerBlockY = min(desiredThreadsPerBlockY, deviceProp.maxThreadsDim[1]);

        dim3 dimBlock(threadsPerBlockX, threadsPerBlockY);
        dim3 dimGrid(blocksX, blocksY);

        Gpu2dDraw<<<dimGrid, dimBlock>>>
        (
            dImgPosX, dImgPosY, dImgWidth, dImgHeight, dImgData, imgAmount, 
            dBuf, bufWidth, bufHeight, bufDpi, backgroundColor
        );
        hipError_t error = hipDeviceSynchronize();
        if (error != 0){
            OutputDebugStringA("GlpaLib ERROR Render.cu - Processing with Cuda failed.\n");
            throw std::runtime_error("Processing with Cuda failed.");
        }

        hipMemcpy(buf, dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD), hipMemcpyDeviceToHost);
    }
    else
    {
        hipMemcpy(buf, dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD), hipMemcpyDeviceToHost);
    }
}

Glpa::Render3d::Render3d()
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "Constructor");
}

Glpa::Render3d::~Render3d()
{
    Glpa::OutputLog(__FILE__, __LINE__, __FUNCSIG__, Glpa::OUTPUT_TAG_GLPA_LIB, "Destructor");
}

void Glpa::Render3d::run(
    std::unordered_map<std::string, Glpa::SceneObject*> objs, LPDWORD buf, int bufWidth, int bufHeight, int bufDpi
){

}

__global__ void Glpa::Gpu2dDraw
(
    int *imgPosX, int *imgPosY, int* imgWidth, int* imgHeight, LPDWORD *imgData, int imgAmount,
    LPDWORD buf, int bufWidth, int bufHeight, int bufDpi, DWORD background
){
    int x = blockIdx.y * blockDim.y + threadIdx.y;
    int y = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < bufWidth)
    {
        if (y < bufHeight)
        {
            atomicExch((unsigned int*)&buf[x + y * bufWidth * bufDpi], (unsigned int)background);

            int isBackgroundIF = TRUE;
            int isNotBackgroundIF = FALSE;
            for (int i = 0; i < imgAmount; i++)
            {
                int xInImgIF = (x >= imgPosX[i] && x < imgPosX[i] + imgWidth[i]) ? TRUE : FALSE;
                int yInImgIF = (y >= imgPosY[i] && y < imgPosY[i] + imgHeight[i]) ? TRUE : FALSE;

                int writeIF = (xInImgIF == TRUE && yInImgIF == TRUE) ? TRUE : FALSE;

                for (int cb1 = 0; cb1 < writeIF; cb1++)
                {
                    int imgX = x - imgPosX[i];
                    int imgY = y - imgPosY[i];

                    for (int cb2 = 0; cb2 < isNotBackgroundIF; cb2++)
                    {
                        BYTE bufR = (buf[x + y * bufWidth * bufDpi] >> 16) & 0xFF;
                        BYTE bufG = (buf[x + y * bufWidth * bufDpi] >> 8) & 0xFF;
                        BYTE bufB = buf[x + y * bufWidth * bufDpi] & 0xFF;

                        BYTE imgA = (imgData[i][imgX + imgY * imgWidth[i]] >> 24) & 0xFF;
                        BYTE imgR = (imgData[i][imgX + imgY * imgWidth[i]] >> 16) & 0xFF;
                        BYTE imgG = (imgData[i][imgX + imgY * imgWidth[i]] >> 8) & 0xFF;
                        BYTE imgB = imgData[i][imgX + imgY * imgWidth[i]] & 0xFF;

                        float alpha = static_cast<float>(imgA) / 255.0f;
                        float invAlpha = 1.0f - alpha;

                        bufR = static_cast<unsigned char>(alpha * imgR + invAlpha * bufR);
                        bufG = static_cast<unsigned char>(alpha * imgG + invAlpha * bufG);
                        bufB = static_cast<unsigned char>(alpha * imgB + invAlpha * bufB);

                        DWORD newColor = (1 << 24) | (bufR << 16) | (bufG << 8) | bufB;
                        atomicExch((unsigned int*)&buf[x + y * bufWidth * bufDpi], (unsigned int)newColor);
                    }

                    for (int cb2 = 0; cb2 < isBackgroundIF; cb2++)
                    {
                        atomicExch((unsigned int*)&buf[x + y * bufWidth * bufDpi], (unsigned int)imgData[i][imgX + imgY * imgWidth[i]]);
                        isBackgroundIF = FALSE;
                        isNotBackgroundIF = TRUE;
                    }

                }
            }
        }
    }
}
