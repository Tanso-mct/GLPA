#include "hip/hip_runtime.h"
#include "render.cuh"

Render::Render()
{
    hMtCamTransRot = std::vector<float>(16);
    hMtCamRot = std::vector<float>(16);

    hCamViewAngleCos = std::vector<float>(2);

}

__global__ void glpaGpuPrepareObj(
    int objSize,
    float* objWVs,
    float* mtCamTransRot,
    float camNearZ,
    float camFarZ,
    float* camViewAngleCos,
    int* result
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < objSize)
    {
        int objRectStatus = 0;
        float objRectOrigin[3];
        float objRectOpposite[3];
        for (int objWvsI = 0; objWvsI < 8; objWvsI++)
        {
            float vec3d[3] = {objWVs[i*8*3 + objWvsI*3 + AX], objWVs[i*8*3 + objWvsI*3 + AY], objWVs[i*8*3 + objWvsI*3 + AZ]};

            float camObjVs[3] = {
                vec3d[AX] * mtCamTransRot[0] + vec3d[AY] * mtCamTransRot[1] + vec3d[AZ] * mtCamTransRot[2] + 1 * mtCamTransRot[3],
                vec3d[AX] * mtCamTransRot[4] + vec3d[AY] * mtCamTransRot[5] + vec3d[AZ] * mtCamTransRot[6] + 1 * mtCamTransRot[7],
                vec3d[AX] * mtCamTransRot[8] + vec3d[AY] * mtCamTransRot[9] + vec3d[AZ] * mtCamTransRot[10] + 1 * mtCamTransRot[11]
            };

            int objRectStatusIF = (objRectStatus > 0) ? TRUE : FALSE;

            objRectOrigin[AX] = (objRectStatusIF == FALSE) ? camObjVs[AX] : (camObjVs[AX] < objRectOrigin[AX]) ? camObjVs[AX] : objRectOrigin[AX];
            objRectOrigin[AY] = (objRectStatusIF == FALSE) ? camObjVs[AY] : (camObjVs[AY] < objRectOrigin[AY]) ? camObjVs[AY] : objRectOrigin[AY];
            objRectOrigin[AZ] = (objRectStatusIF == FALSE) ? camObjVs[AZ] : (camObjVs[AZ] > objRectOrigin[AZ]) ? camObjVs[AZ] : objRectOrigin[AZ];

            objRectOpposite[AX] = (objRectStatusIF == FALSE) ? camObjVs[AX] : (camObjVs[AX] > objRectOpposite[AX]) ? camObjVs[AX] : objRectOpposite[AX];
            objRectOpposite[AY] = (objRectStatusIF == FALSE) ? camObjVs[AY] : (camObjVs[AY] > objRectOpposite[AY]) ? camObjVs[AY] : objRectOpposite[AY];
            objRectOpposite[AZ] = (objRectStatusIF == FALSE) ? camObjVs[AZ] : (camObjVs[AZ] < objRectOpposite[AZ]) ? camObjVs[AZ] : objRectOpposite[AZ];

            objRectStatus += 1;

        }

        float objOppositeVs[12] = {
            objRectOrigin[AX], 0, objRectOpposite[AZ],
            objRectOpposite[AX], 0, objRectOpposite[AZ],
            0, objRectOrigin[AY], objRectOpposite[AZ],
            0, objRectOpposite[AY], objRectOpposite[AZ]
        };


        float zVec[3] = {0, 0, -1};
        float vecsCos[4];

        for (int aryI = 0; aryI < 4; aryI++)
        {
            float calcObjOppositeV[3] = {
                objOppositeVs[aryI*3 + AX],
                objOppositeVs[aryI*3 + AY],
                objOppositeVs[aryI*3 + AZ]
            };

            VEC_GET_VECS_COS(zVec, calcObjOppositeV, vecsCos[aryI]);
        }

        int objZInIF = (objRectOrigin[AZ] >= -camFarZ && objRectOpposite[AZ] <= -camNearZ) ? TRUE : FALSE;
        int objXzInIF = (vecsCos[0] >= camViewAngleCos[AX] || vecsCos[1] >= camViewAngleCos[AX]) ? TRUE : FALSE;
        int objYzInIF = (vecsCos[2] >= camViewAngleCos[AY] || vecsCos[3] >= camViewAngleCos[AY]) ? TRUE : FALSE;

        int objInIF = (objZInIF == TRUE && objXzInIF == TRUE && objYzInIF == TRUE) ? i + 1 : 0;

        result[objInIF] = TRUE;
    }
}

void Render::prepareObjs(std::unordered_map<std::wstring, Object> sObj, Camera cam)
{
    std::vector<float> hObjWvs;

    for (auto obj : sObj)
    {
        for (int i = 0; i < 8; i++)
        {
            hObjWvs.push_back(obj.second.range.wVertex[i].x / CALC_SCALE);
            hObjWvs.push_back(obj.second.range.wVertex[i].y / CALC_SCALE);
            hObjWvs.push_back(obj.second.range.wVertex[i].z / CALC_SCALE);
        }
    }

    float* dObjWvs;
    hipMalloc((void**)&dObjWvs, sizeof(float)*hObjWvs.size());
    hipMemcpy(dObjWvs, hObjWvs.data(), sizeof(float)*hObjWvs.size(), hipMemcpyHostToDevice);

    hMtCamTransRot[0] = cos(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.y));
    hMtCamTransRot[1] = cos(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * sin(RAD(-cam.rotAngle.x)) + -sin(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.x));
    hMtCamTransRot[2] = cos(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * cos(RAD(-cam.rotAngle.x)) + -sin(RAD(-cam.rotAngle.z)) * -sin(RAD(-cam.rotAngle.x));
    hMtCamTransRot[3] = -cam.wPos.x / CALC_SCALE;
    hMtCamTransRot[4] = sin(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.y));
    hMtCamTransRot[5] = sin(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * sin(RAD(-cam.rotAngle.x)) + cos(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.x));
    hMtCamTransRot[6] = sin(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * cos(RAD(-cam.rotAngle.x)) + cos(RAD(-cam.rotAngle.z)) * -sin(RAD(-cam.rotAngle.x));
    hMtCamTransRot[7] = -cam.wPos.y / CALC_SCALE;
    hMtCamTransRot[8] = -sin(RAD(-cam.rotAngle.y));
    hMtCamTransRot[9] = cos(RAD(-cam.rotAngle.y)) * sin(RAD(-cam.rotAngle.x));
    hMtCamTransRot[10] = cos(RAD(-cam.rotAngle.y)) * cos(RAD(-cam.rotAngle.x));
    hMtCamTransRot[11] = -cam.wPos.z / CALC_SCALE;
    hMtCamTransRot[12] = 0;
    hMtCamTransRot[13] = 0;
    hMtCamTransRot[14] = 0;
    hMtCamTransRot[15] = 1;

    float* dMtCamTransRot;
    hipMalloc((void**)&dMtCamTransRot, sizeof(float)*hMtCamTransRot.size());
    hipMemcpy(dMtCamTransRot, hMtCamTransRot.data(), sizeof(float)*hMtCamTransRot.size(), hipMemcpyHostToDevice);

    hCamViewAngleCos[AX] = cam.viewAngleCos.x;
    hCamViewAngleCos[AY] = cam.viewAngleCos.y;

    float* dCamViewAngleCos;
    hipMalloc((void**)&dCamViewAngleCos, sizeof(float)*hCamViewAngleCos.size());
    hipMemcpy(dCamViewAngleCos, hCamViewAngleCos.data(), sizeof(float)*hCamViewAngleCos.size(), hipMemcpyHostToDevice);


    hObjInJudgeAry = new int[sObj.size() + 1];
    std::fill(hObjInJudgeAry, hObjInJudgeAry + sObj.size() + 1, FALSE); 

    int* dObjInJudgeAry;
    hipMalloc((void**)&dObjInJudgeAry, sizeof(int)*(sObj.size() + 1));


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int dataSize = sObj.size();
    int desiredThreadsPerBlock = 256;

    int blocks = (dataSize + desiredThreadsPerBlock - 1) / desiredThreadsPerBlock;
    int threadsPerBlock = std::min(desiredThreadsPerBlock, deviceProp.maxThreadsPerBlock);

    dim3 dimBlock(threadsPerBlock);
    dim3 dimGrid(blocks);

    glpaGpuPrepareObj<<<dimGrid, dimBlock>>>
    (
        sObj.size(),
        dObjWvs,
        dMtCamTransRot,
        static_cast<float>(cam.nearZ) / CALC_SCALE,
        static_cast<float>(cam.farZ) / CALC_SCALE,
        dCamViewAngleCos,
        dObjInJudgeAry
    );
    hipDeviceSynchronize();

    hipError_t error = hipGetLastError();
    if (error != 0)
    {
        throw std::runtime_error(ERROR_VECTOR_CUDA_ERROR);
    }

    hipMemcpy(hObjInJudgeAry, dObjInJudgeAry, sizeof(int)*(sObj.size() + 1), hipMemcpyDeviceToHost);

    hipFree(dObjWvs);
    hipFree(dMtCamTransRot);
    hipFree(dCamViewAngleCos);
    hipFree(dObjInJudgeAry);

    hCamViewAngleCos = std::vector<float>(2);

}

__global__ void glpaGpuRender(
    float* polyVs,
    float* polyNs,
    int polyAmount,
    float* mtCamTransRot,
    float* mtCamRot,
    float camFarZ,
    float camNearZ,
    float* camViewAngleCos,
    float* viewVolumeVs,
    float* viewVolumeNs,
    float* nearScSize,
    float* scPixelSize,
    float* result
){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < polyAmount)
    {
        float vec3d[3] = {0};

        vec3d[AX] = polyVs[i*9 + AX];
        vec3d[AY] = polyVs[i*9 + AY];
        vec3d[AZ] = polyVs[i*9 + AZ];
        float cnvtPolyV1[3];
        MT_PRODUCT_4X4_VEC3D(mtCamTransRot, vec3d, cnvtPolyV1);

        vec3d[AX] = polyVs[i*9 + 3 + AX];
        vec3d[AY] = polyVs[i*9 + 3 + AY];
        vec3d[AZ] = polyVs[i*9 + 3 + AZ];
        float cnvtPolyV2[3];
        MT_PRODUCT_4X4_VEC3D(mtCamTransRot, vec3d, cnvtPolyV2);

        vec3d[AX] = polyVs[i*9 + 6 + AX];
        vec3d[AY] = polyVs[i*9 + 6 + AY];
        vec3d[AZ] = polyVs[i*9 + 6 + AZ];
        float cnvtPolyV3[3];
        MT_PRODUCT_4X4_VEC3D(mtCamTransRot, vec3d, cnvtPolyV3);

        vec3d[AX] = polyNs[i*3 + AX];
        vec3d[AY] = polyNs[i*3 + AY];
        vec3d[AZ] = polyNs[i*3 + AZ];
        float cnvtPolyN[3];
        MT_PRODUCT_4X4_VEC3D(mtCamRot, vec3d, cnvtPolyN);

        float polyVxPolyNDotCos;
        VEC_GET_VECS_COS(cnvtPolyN, cnvtPolyV1, polyVxPolyNDotCos);
        
        int polyBilateralIF = (polyVxPolyNDotCos <= 0) ? TRUE : FALSE;

        for (int conditionalBranch = 0; conditionalBranch < polyBilateralIF; conditionalBranch++)
        {
            int polyV1InIF;
            int polyV2InIF;
            int polyV3InIF;
            JUDGE_POLY_V_IN_VIEW_VOLUME(cnvtPolyV1, camFarZ, camNearZ, camViewAngleCos, polyV1InIF);
            JUDGE_POLY_V_IN_VIEW_VOLUME(cnvtPolyV2, camFarZ, camNearZ, camViewAngleCos, polyV2InIF);
            JUDGE_POLY_V_IN_VIEW_VOLUME(cnvtPolyV3, camFarZ, camNearZ, camViewAngleCos, polyV3InIF);

            int noVsInIF = (polyV1InIF == FALSE && polyV2InIF == FALSE && polyV3InIF == FALSE) ? TRUE : FALSE;

            int shapeCnvtIF = (polyV1InIF + polyV2InIF + polyV3InIF != 3) ? TRUE : FALSE;

            int polyInIF = (polyV1InIF == TRUE || polyV2InIF == TRUE || polyV3InIF == TRUE) ? TRUE : FALSE;
            for (int conditionalBranch2 = 0; conditionalBranch2 < noVsInIF; conditionalBranch2++)
            {
                float polyRectOrigin[3] = {cnvtPolyV1[AX], cnvtPolyV1[AY], cnvtPolyV1[AZ]};
                float polyRectOpposite[3] = {cnvtPolyV1[AX], cnvtPolyV1[AY], cnvtPolyV1[AZ]};

                polyRectOrigin[AX] = (cnvtPolyV2[AX] < polyRectOrigin[AX]) ? cnvtPolyV2[AX] : polyRectOrigin[AX];
                polyRectOrigin[AY] = (cnvtPolyV2[AY] < polyRectOrigin[AY]) ? cnvtPolyV2[AY] : polyRectOrigin[AY];
                polyRectOrigin[AZ] = (cnvtPolyV2[AZ] > polyRectOrigin[AZ]) ? cnvtPolyV2[AZ] : polyRectOrigin[AZ];

                polyRectOpposite[AX] = (cnvtPolyV2[AX] > polyRectOpposite[AX]) ? cnvtPolyV2[AX] : polyRectOpposite[AX];
                polyRectOpposite[AY] = (cnvtPolyV2[AY] > polyRectOpposite[AY]) ? cnvtPolyV2[AY] : polyRectOpposite[AY];
                polyRectOpposite[AZ] = (cnvtPolyV2[AZ] < polyRectOpposite[AZ]) ? cnvtPolyV2[AZ] : polyRectOpposite[AZ];

                polyRectOrigin[AX] = (cnvtPolyV3[AX] < polyRectOrigin[AX]) ? cnvtPolyV3[AX] : polyRectOrigin[AX];
                polyRectOrigin[AY] = (cnvtPolyV3[AY] < polyRectOrigin[AY]) ? cnvtPolyV3[AY] : polyRectOrigin[AY];
                polyRectOrigin[AZ] = (cnvtPolyV3[AZ] > polyRectOrigin[AZ]) ? cnvtPolyV3[AZ] : polyRectOrigin[AZ];

                polyRectOpposite[AX] = (cnvtPolyV3[AX] > polyRectOpposite[AX]) ? cnvtPolyV3[AX] : polyRectOpposite[AX];
                polyRectOpposite[AY] = (cnvtPolyV3[AY] > polyRectOpposite[AY]) ? cnvtPolyV3[AY] : polyRectOpposite[AY];
                polyRectOpposite[AZ] = (cnvtPolyV3[AZ] < polyRectOpposite[AZ]) ? cnvtPolyV3[AZ] : polyRectOpposite[AZ];

                // TODO: 3 and 4 are different from the source. This may be the cause of the bug, so please check.
                float polyRectOppositeSideVs[12] = {
                    polyRectOrigin[AX], 0,  polyRectOpposite[AZ],
                    polyRectOpposite[AX], 0, polyRectOpposite[AZ],
                    0, polyRectOrigin[AY], polyRectOpposite[AZ],
                    0, polyRectOpposite[AY], polyRectOpposite[AZ]
                };

                float zVec[3] = {0, 0, -1};
                float vecsCos[4];

                for (int aryI = 0; aryI < 4; aryI++)
                {
                    float calcObjOppositeV[3] = {
                        polyRectOppositeSideVs[aryI*3 + AX],
                        polyRectOppositeSideVs[aryI*3 + AY],
                        polyRectOppositeSideVs[aryI*3 + AZ]
                    };

                    VEC_GET_VECS_COS(zVec, calcObjOppositeV, vecsCos[aryI]);
                }

                int polyZInIF = (polyRectOrigin[AZ] >= -camFarZ && polyRectOpposite[AZ] <= -camNearZ) ? TRUE : FALSE;
                int polyXzInIF = (vecsCos[0] >= camViewAngleCos[AX] || vecsCos[1] >= camViewAngleCos[AX]) ? TRUE : FALSE;
                int polyYzInIF = (vecsCos[2] >= camViewAngleCos[AY] || vecsCos[3] >= camViewAngleCos[AY]) ? TRUE : FALSE;

                polyInIF = (polyZInIF == TRUE && polyXzInIF == TRUE && polyYzInIF == TRUE) ? TRUE : FALSE;
            }

            for(int conditionalBranch2 = 0; conditionalBranch2 < polyInIF; conditionalBranch2++)
            {
                int vvFaceI[6] = {
                    RECT_FRONT_TOP_LEFT,
                    RECT_FRONT_TOP_LEFT,
                    RECT_BACK_BOTTOM_RIGHT,
                    RECT_FRONT_TOP_LEFT,
                    RECT_BACK_BOTTOM_RIGHT,
                    RECT_BACK_BOTTOM_RIGHT
                };

                int vvFaceVsI[24] = {
                    VIEWVOLUME_TOP_V1, VIEWVOLUME_TOP_V2, VIEWVOLUME_TOP_V3, VIEWVOLUME_TOP_V4,
                    VIEWVOLUME_FRONT_V1, VIEWVOLUME_FRONT_V2, VIEWVOLUME_FRONT_V3, VIEWVOLUME_FRONT_V4,
                    VIEWVOLUME_RIGHT_V1, VIEWVOLUME_RIGHT_V2, VIEWVOLUME_RIGHT_V3, VIEWVOLUME_RIGHT_V4,
                    VIEWVOLUME_LEFT_V1, VIEWVOLUME_LEFT_V2, VIEWVOLUME_LEFT_V3, VIEWVOLUME_LEFT_V4,
                    VIEWVOLUME_BACK_V1, VIEWVOLUME_BACK_V2, VIEWVOLUME_BACK_V3, VIEWVOLUME_BACK_V4,
                    VIEWVOLUME_BOTTOM_V1, VIEWVOLUME_BOTTOM_V2, VIEWVOLUME_BOTTOM_V3, VIEWVOLUME_BOTTOM_V4
                };

                int vvLineVI[24] = {
                    RECT_L1_STARTV, RECT_L1_ENDV,
                    RECT_L2_STARTV, RECT_L2_ENDV,
                    RECT_L3_STARTV, RECT_L3_ENDV,
                    RECT_L4_STARTV, RECT_L4_ENDV,
                    RECT_L5_STARTV, RECT_L5_ENDV,
                    RECT_L6_STARTV, RECT_L6_ENDV,
                    RECT_L7_STARTV, RECT_L7_ENDV,
                    RECT_L8_STARTV, RECT_L8_ENDV,
                    RECT_L9_STARTV, RECT_L9_ENDV,
                    RECT_L10_STARTV, RECT_L10_ENDV,
                    RECT_L11_STARTV, RECT_L11_ENDV,
                    RECT_L12_STARTV, RECT_L12_ENDV
                };

                int inxtnAmount = 0;

                // float pixelVs[12 * 3 * 3 + 3*3 + 3*3] = {-2};
                int pixelVsSize = 12 * 3 * 3 + 3*3 + 3*3;
                int targetIndex = 0;

                for (int roopLineI = 0; roopLineI < 12; roopLineI++)
                {
                    float polyFaceDot[2];
                    CALC_POLY_FACE_DOT(polyFaceDot, viewVolumeVs, vvLineVI[roopLineI*2], vvLineVI[roopLineI*2 + 1], cnvtPolyV1, cnvtPolyN);

                    // ERROR DEBUG
                    do
                    {
                        int vOnFaceIF = (polyFaceDot[0] == 0) ? 1 : 0;
                        for (int conditionalBranch3; conditionalBranch3 < vOnFaceIF; conditionalBranch3++)
                        {
                            float inxtn[3] = {viewVolumeVs[vvLineVI[roopLineI * 2] * 3 + 0], viewVolumeVs[vvLineVI[roopLineI * 2] * 3 + 1], viewVolumeVs[vvLineVI[roopLineI * 2] * 3 + 2]};
                            float vecCos[6];
                            do
                            {
                                vecCos[0] = ((cnvtPolyV2[0] - cnvtPolyV1[0]) * (inxtn[0] - cnvtPolyV1[0]) + (cnvtPolyV2[1] - cnvtPolyV1[1]) * (inxtn[1] - cnvtPolyV1[1]) + (cnvtPolyV2[2] - cnvtPolyV1[2]) * (inxtn[2] - cnvtPolyV1[2])) / (sqrt((cnvtPolyV2[0] - cnvtPolyV1[0]) * (cnvtPolyV2[0] - cnvtPolyV1[0]) + (cnvtPolyV2[1] - cnvtPolyV1[1]) * (cnvtPolyV2[1] - cnvtPolyV1[1]) + (cnvtPolyV2[2] - cnvtPolyV1[2]) * (cnvtPolyV2[2] - cnvtPolyV1[2])) * sqrt((inxtn[0] - cnvtPolyV1[0]) * (inxtn[0] - cnvtPolyV1[0]) + (inxtn[1] - cnvtPolyV1[1]) * (inxtn[1] - cnvtPolyV1[1]) + (inxtn[2] - cnvtPolyV1[2]) * (inxtn[2] - cnvtPolyV1[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[1] = ((cnvtPolyV2[0] - cnvtPolyV1[0]) * (cnvtPolyV3[0] - cnvtPolyV1[0]) + (cnvtPolyV2[1] - cnvtPolyV1[1]) * (cnvtPolyV3[1] - cnvtPolyV1[1]) + (cnvtPolyV2[2] - cnvtPolyV1[2]) * (cnvtPolyV3[2] - cnvtPolyV1[2])) / (sqrt((cnvtPolyV2[0] - cnvtPolyV1[0]) * (cnvtPolyV2[0] - cnvtPolyV1[0]) + (cnvtPolyV2[1] - cnvtPolyV1[1]) * (cnvtPolyV2[1] - cnvtPolyV1[1]) + (cnvtPolyV2[2] - cnvtPolyV1[2]) * (cnvtPolyV2[2] - cnvtPolyV1[2])) * sqrt((cnvtPolyV3[0] - cnvtPolyV1[0]) * (cnvtPolyV3[0] - cnvtPolyV1[0]) + (cnvtPolyV3[1] - cnvtPolyV1[1]) * (cnvtPolyV3[1] - cnvtPolyV1[1]) + (cnvtPolyV3[2] - cnvtPolyV1[2]) * (cnvtPolyV3[2] - cnvtPolyV1[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[2] = ((cnvtPolyV3[0] - cnvtPolyV2[0]) * (inxtn[0] - cnvtPolyV2[0]) + (cnvtPolyV3[1] - cnvtPolyV2[1]) * (inxtn[1] - cnvtPolyV2[1]) + (cnvtPolyV3[2] - cnvtPolyV2[2]) * (inxtn[2] - cnvtPolyV2[2])) / (sqrt((cnvtPolyV3[0] - cnvtPolyV2[0]) * (cnvtPolyV3[0] - cnvtPolyV2[0]) + (cnvtPolyV3[1] - cnvtPolyV2[1]) * (cnvtPolyV3[1] - cnvtPolyV2[1]) + (cnvtPolyV3[2] - cnvtPolyV2[2]) * (cnvtPolyV3[2] - cnvtPolyV2[2])) * sqrt((inxtn[0] - cnvtPolyV2[0]) * (inxtn[0] - cnvtPolyV2[0]) + (inxtn[1] - cnvtPolyV2[1]) * (inxtn[1] - cnvtPolyV2[1]) + (inxtn[2] - cnvtPolyV2[2]) * (inxtn[2] - cnvtPolyV2[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[3] = ((cnvtPolyV3[0] - cnvtPolyV2[0]) * (cnvtPolyV1[0] - cnvtPolyV2[0]) + (cnvtPolyV3[1] - cnvtPolyV2[1]) * (cnvtPolyV1[1] - cnvtPolyV2[1]) + (cnvtPolyV3[2] - cnvtPolyV2[2]) * (cnvtPolyV1[2] - cnvtPolyV2[2])) / (sqrt((cnvtPolyV3[0] - cnvtPolyV2[0]) * (cnvtPolyV3[0] - cnvtPolyV2[0]) + (cnvtPolyV3[1] - cnvtPolyV2[1]) * (cnvtPolyV3[1] - cnvtPolyV2[1]) + (cnvtPolyV3[2] - cnvtPolyV2[2]) * (cnvtPolyV3[2] - cnvtPolyV2[2])) * sqrt((cnvtPolyV1[0] - cnvtPolyV2[0]) * (cnvtPolyV1[0] - cnvtPolyV2[0]) + (cnvtPolyV1[1] - cnvtPolyV2[1]) * (cnvtPolyV1[1] - cnvtPolyV2[1]) + (cnvtPolyV1[2] - cnvtPolyV2[2]) * (cnvtPolyV1[2] - cnvtPolyV2[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[4] = ((cnvtPolyV1[0] - cnvtPolyV3[0]) * (inxtn[0] - cnvtPolyV3[0]) + (cnvtPolyV1[1] - cnvtPolyV3[1]) * (inxtn[1] - cnvtPolyV3[1]) + (cnvtPolyV1[2] - cnvtPolyV3[2]) * (inxtn[2] - cnvtPolyV3[2])) / (sqrt((cnvtPolyV1[0] - cnvtPolyV3[0]) * (cnvtPolyV1[0] - cnvtPolyV3[0]) + (cnvtPolyV1[1] - cnvtPolyV3[1]) * (cnvtPolyV1[1] - cnvtPolyV3[1]) + (cnvtPolyV1[2] - cnvtPolyV3[2]) * (cnvtPolyV1[2] - cnvtPolyV3[2])) * sqrt((inxtn[0] - cnvtPolyV3[0]) * (inxtn[0] - cnvtPolyV3[0]) + (inxtn[1] - cnvtPolyV3[1]) * (inxtn[1] - cnvtPolyV3[1]) + (inxtn[2] - cnvtPolyV3[2]) * (inxtn[2] - cnvtPolyV3[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[5] = ((cnvtPolyV1[0] - cnvtPolyV3[0]) * (cnvtPolyV2[0] - cnvtPolyV3[0]) + (cnvtPolyV1[1] - cnvtPolyV3[1]) * (cnvtPolyV2[1] - cnvtPolyV3[1]) + (cnvtPolyV1[2] - cnvtPolyV3[2]) * (cnvtPolyV2[2] - cnvtPolyV3[2])) / (sqrt((cnvtPolyV1[0] - cnvtPolyV3[0]) * (cnvtPolyV1[0] - cnvtPolyV3[0]) + (cnvtPolyV1[1] - cnvtPolyV3[1]) * (cnvtPolyV1[1] - cnvtPolyV3[1]) + (cnvtPolyV1[2] - cnvtPolyV3[2]) * (cnvtPolyV1[2] - cnvtPolyV3[2])) * sqrt((cnvtPolyV2[0] - cnvtPolyV3[0]) * (cnvtPolyV2[0] - cnvtPolyV3[0]) + (cnvtPolyV2[1] - cnvtPolyV3[1]) * (cnvtPolyV2[1] - cnvtPolyV3[1]) + (cnvtPolyV2[2] - cnvtPolyV3[2]) * (cnvtPolyV2[2] - cnvtPolyV3[2])));
                            } while (0);
                            ;
                            int inxtnInPolyFaceIF = (vecCos[0] >= vecCos[1] && vecCos[2] >= vecCos[3] && vecCos[4] >= vecCos[5]) ? 1 : 0;
                            for (int conditionalBranch4 = 0; conditionalBranch4 < inxtnInPolyFaceIF; conditionalBranch4++)
                            {
                                // do
                                // {
                                //     result[(i * pixelVsSize + targetIndex) * 3 + 0] = std::round((((viewVolumeVs[vvLineVI[roopLineI * 2] * 3 + 0] * -camNearZ / viewVolumeVs[vvLineVI[roopLineI * 2] * 3 + 2]) + nearScSize[0] / 2) / (nearScSize[0])) * scPixelSize[0]);
                                // } while (0);
                                // ;
                                // do
                                // {
                                //     result[(i * pixelVsSize + targetIndex) * 3 + 1] = std::round(scPixelSize[1] - (((viewVolumeVs[vvLineVI[roopLineI * 2] * 3 + 1] * -camNearZ / viewVolumeVs[vvLineVI[roopLineI * 2] * 3 + 2]) + nearScSize[1] / 2) / (nearScSize[1])) * scPixelSize[1]);
                                // } while (0);
                                // ;
                                // result[(i * pixelVsSize + targetIndex) * 3 + 2] = viewVolumeVs[vvLineVI[roopLineI * 2] * 3 + 2];
                                // targetIndex++;
                            }
                        }
                    } while (0);

                    // infinite roop error
                    do
                    {
                        int vOnFaceIF = (polyFaceDot[1] == 0) ? 1 : 0;
                        for (int conditionalBranch3; conditionalBranch3 < vOnFaceIF; conditionalBranch3++)
                        {
                            float inxtn[3] = {viewVolumeVs[vvLineVI[roopLineI * 2 + 1] * 3 + 0], viewVolumeVs[vvLineVI[roopLineI * 2 + 1] * 3 + 1], viewVolumeVs[vvLineVI[roopLineI * 2 + 1] * 3 + 2]};
                            float vecCos[6];
                            do
                            {
                                vecCos[0] = ((cnvtPolyV2[0] - cnvtPolyV1[0]) * (inxtn[0] - cnvtPolyV1[0]) + (cnvtPolyV2[1] - cnvtPolyV1[1]) * (inxtn[1] - cnvtPolyV1[1]) + (cnvtPolyV2[2] - cnvtPolyV1[2]) * (inxtn[2] - cnvtPolyV1[2])) / (sqrt((cnvtPolyV2[0] - cnvtPolyV1[0]) * (cnvtPolyV2[0] - cnvtPolyV1[0]) + (cnvtPolyV2[1] - cnvtPolyV1[1]) * (cnvtPolyV2[1] - cnvtPolyV1[1]) + (cnvtPolyV2[2] - cnvtPolyV1[2]) * (cnvtPolyV2[2] - cnvtPolyV1[2])) * sqrt((inxtn[0] - cnvtPolyV1[0]) * (inxtn[0] - cnvtPolyV1[0]) + (inxtn[1] - cnvtPolyV1[1]) * (inxtn[1] - cnvtPolyV1[1]) + (inxtn[2] - cnvtPolyV1[2]) * (inxtn[2] - cnvtPolyV1[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[1] = ((cnvtPolyV2[0] - cnvtPolyV1[0]) * (cnvtPolyV3[0] - cnvtPolyV1[0]) + (cnvtPolyV2[1] - cnvtPolyV1[1]) * (cnvtPolyV3[1] - cnvtPolyV1[1]) + (cnvtPolyV2[2] - cnvtPolyV1[2]) * (cnvtPolyV3[2] - cnvtPolyV1[2])) / (sqrt((cnvtPolyV2[0] - cnvtPolyV1[0]) * (cnvtPolyV2[0] - cnvtPolyV1[0]) + (cnvtPolyV2[1] - cnvtPolyV1[1]) * (cnvtPolyV2[1] - cnvtPolyV1[1]) + (cnvtPolyV2[2] - cnvtPolyV1[2]) * (cnvtPolyV2[2] - cnvtPolyV1[2])) * sqrt((cnvtPolyV3[0] - cnvtPolyV1[0]) * (cnvtPolyV3[0] - cnvtPolyV1[0]) + (cnvtPolyV3[1] - cnvtPolyV1[1]) * (cnvtPolyV3[1] - cnvtPolyV1[1]) + (cnvtPolyV3[2] - cnvtPolyV1[2]) * (cnvtPolyV3[2] - cnvtPolyV1[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[2] = ((cnvtPolyV3[0] - cnvtPolyV2[0]) * (inxtn[0] - cnvtPolyV2[0]) + (cnvtPolyV3[1] - cnvtPolyV2[1]) * (inxtn[1] - cnvtPolyV2[1]) + (cnvtPolyV3[2] - cnvtPolyV2[2]) * (inxtn[2] - cnvtPolyV2[2])) / (sqrt((cnvtPolyV3[0] - cnvtPolyV2[0]) * (cnvtPolyV3[0] - cnvtPolyV2[0]) + (cnvtPolyV3[1] - cnvtPolyV2[1]) * (cnvtPolyV3[1] - cnvtPolyV2[1]) + (cnvtPolyV3[2] - cnvtPolyV2[2]) * (cnvtPolyV3[2] - cnvtPolyV2[2])) * sqrt((inxtn[0] - cnvtPolyV2[0]) * (inxtn[0] - cnvtPolyV2[0]) + (inxtn[1] - cnvtPolyV2[1]) * (inxtn[1] - cnvtPolyV2[1]) + (inxtn[2] - cnvtPolyV2[2]) * (inxtn[2] - cnvtPolyV2[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[3] = ((cnvtPolyV3[0] - cnvtPolyV2[0]) * (cnvtPolyV1[0] - cnvtPolyV2[0]) + (cnvtPolyV3[1] - cnvtPolyV2[1]) * (cnvtPolyV1[1] - cnvtPolyV2[1]) + (cnvtPolyV3[2] - cnvtPolyV2[2]) * (cnvtPolyV1[2] - cnvtPolyV2[2])) / (sqrt((cnvtPolyV3[0] - cnvtPolyV2[0]) * (cnvtPolyV3[0] - cnvtPolyV2[0]) + (cnvtPolyV3[1] - cnvtPolyV2[1]) * (cnvtPolyV3[1] - cnvtPolyV2[1]) + (cnvtPolyV3[2] - cnvtPolyV2[2]) * (cnvtPolyV3[2] - cnvtPolyV2[2])) * sqrt((cnvtPolyV1[0] - cnvtPolyV2[0]) * (cnvtPolyV1[0] - cnvtPolyV2[0]) + (cnvtPolyV1[1] - cnvtPolyV2[1]) * (cnvtPolyV1[1] - cnvtPolyV2[1]) + (cnvtPolyV1[2] - cnvtPolyV2[2]) * (cnvtPolyV1[2] - cnvtPolyV2[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[4] = ((cnvtPolyV1[0] - cnvtPolyV3[0]) * (inxtn[0] - cnvtPolyV3[0]) + (cnvtPolyV1[1] - cnvtPolyV3[1]) * (inxtn[1] - cnvtPolyV3[1]) + (cnvtPolyV1[2] - cnvtPolyV3[2]) * (inxtn[2] - cnvtPolyV3[2])) / (sqrt((cnvtPolyV1[0] - cnvtPolyV3[0]) * (cnvtPolyV1[0] - cnvtPolyV3[0]) + (cnvtPolyV1[1] - cnvtPolyV3[1]) * (cnvtPolyV1[1] - cnvtPolyV3[1]) + (cnvtPolyV1[2] - cnvtPolyV3[2]) * (cnvtPolyV1[2] - cnvtPolyV3[2])) * sqrt((inxtn[0] - cnvtPolyV3[0]) * (inxtn[0] - cnvtPolyV3[0]) + (inxtn[1] - cnvtPolyV3[1]) * (inxtn[1] - cnvtPolyV3[1]) + (inxtn[2] - cnvtPolyV3[2]) * (inxtn[2] - cnvtPolyV3[2])));
                            } while (0);
                            ;
                            do
                            {
                                vecCos[5] = ((cnvtPolyV1[0] - cnvtPolyV3[0]) * (cnvtPolyV2[0] - cnvtPolyV3[0]) + (cnvtPolyV1[1] - cnvtPolyV3[1]) * (cnvtPolyV2[1] - cnvtPolyV3[1]) + (cnvtPolyV1[2] - cnvtPolyV3[2]) * (cnvtPolyV2[2] - cnvtPolyV3[2])) / (sqrt((cnvtPolyV1[0] - cnvtPolyV3[0]) * (cnvtPolyV1[0] - cnvtPolyV3[0]) + (cnvtPolyV1[1] - cnvtPolyV3[1]) * (cnvtPolyV1[1] - cnvtPolyV3[1]) + (cnvtPolyV1[2] - cnvtPolyV3[2]) * (cnvtPolyV1[2] - cnvtPolyV3[2])) * sqrt((cnvtPolyV2[0] - cnvtPolyV3[0]) * (cnvtPolyV2[0] - cnvtPolyV3[0]) + (cnvtPolyV2[1] - cnvtPolyV3[1]) * (cnvtPolyV2[1] - cnvtPolyV3[1]) + (cnvtPolyV2[2] - cnvtPolyV3[2]) * (cnvtPolyV2[2] - cnvtPolyV3[2])));
                            } while (0);
                            ;
                            int inxtnInPolyFaceIF = (vecCos[0] >= vecCos[1] && vecCos[2] >= vecCos[3] && vecCos[4] >= vecCos[5]) ? 1 : 0;
                            for (int conditionalBranch4 = 0; conditionalBranch4 < inxtnInPolyFaceIF; conditionalBranch4++)
                            {
                                // do
                                // {
                                //     result[(i * pixelVsSize + targetIndex) * 3 + 0] = std::round((((viewVolumeVs[vvLineVI[roopLineI * 2 + 1] * 3 + 0] * -camNearZ / viewVolumeVs[vvLineVI[roopLineI * 2 + 1] * 3 + 2]) + nearScSize[0] / 2) / (nearScSize[0])) * scPixelSize[0]);
                                // } while (0);
                                // ;
                                // do
                                // {
                                //     result[(i * pixelVsSize + targetIndex) * 3 + 1] = std::round(scPixelSize[1] - (((viewVolumeVs[vvLineVI[roopLineI * 2 + 1] * 3 + 1] * -camNearZ / viewVolumeVs[vvLineVI[roopLineI * 2 + 1] * 3 + 2]) + nearScSize[1] / 2) / (nearScSize[1])) * scPixelSize[1]);
                                // } while (0);
                                // ;
                                // result[(i * pixelVsSize + targetIndex) * 3 + 2] = viewVolumeVs[vvLineVI[roopLineI * 2 + 1] * 3 + 2];
                                // targetIndex++;
                            }
                        }
                    } while (0);

                    // JUDGE_V_ON_POLY_FACE(
                    //     result, i*pixelVsSize + targetIndex, targetIndex, polyFaceDot[0], roopLineI, viewVolumeVs, vvLineVI[roopLineI*2], 
                    //     cnvtPolyV1, cnvtPolyV2, cnvtPolyV3, camNearZ, nearScSize, scPixelSize
                    // );

                    // JUDGE_V_ON_POLY_FACE(
                    //     result, i*pixelVsSize + targetIndex, targetIndex, polyFaceDot[1], roopLineI, viewVolumeVs, vvLineVI[roopLineI*2 + 1], 
                    //     cnvtPolyV1, cnvtPolyV2, cnvtPolyV3, camNearZ, nearScSize, scPixelSize
                    // );

                    // GET_POLY_ON_FACE_INXTN(
                    //     result, i*pixelVsSize + targetIndex, targetIndex, polyFaceDot, viewVolumeNs, vvLineVI[roopLineI*2], vvLineVI[roopLineI*2 + 1], 
                    //     cnvtPolyV1, cnvtPolyV2, cnvtPolyV3, camNearZ, nearScSize, scPixelSize
                    // );

                    
                }

                // for (int roopFaceI = 0; roopFaceI < 6; roopFaceI++)
                // {
                //     float vvFaceDot[2];
                //     CALC_VV_FACE_DOT(vvFaceDot, cnvtPolyV1, cnvtPolyV2, viewVolumeVs, vvFaceI[roopFaceI], viewVolumeNs, roopFaceI);
                //     JUDGE_V_ON_VV_FACE(
                //         result, i*pixelVsSize + targetIndex, targetIndex, vvFaceDot[0], cnvtPolyV1, roopFaceI, 
                //         viewVolumeVs, vvFaceVsI, camNearZ, nearScSize, scPixelSize
                //     );
                //     JUDGE_V_ON_VV_FACE(
                //         result, i*pixelVsSize + targetIndex, targetIndex, vvFaceDot[1], cnvtPolyV2, roopFaceI, 
                //         viewVolumeVs, vvFaceVsI, camNearZ, nearScSize, scPixelSize
                //     );
                //     GET_POLY_ON_LINE_INXTN(
                //         result, i*pixelVsSize + targetIndex, targetIndex, cnvtPolyV1, cnvtPolyV2, vvFaceDot, 
                //         viewVolumeVs, vvFaceVsI, roopFaceI, camNearZ, nearScSize, scPixelSize
                //     );

                //     CALC_VV_FACE_DOT(vvFaceDot, cnvtPolyV2, cnvtPolyV3, viewVolumeVs, vvFaceI[roopFaceI], viewVolumeNs, roopFaceI);
                //     JUDGE_V_ON_VV_FACE(
                //         result, i*pixelVsSize + targetIndex, targetIndex, vvFaceDot[1], cnvtPolyV3, roopFaceI, 
                //         viewVolumeVs, vvFaceVsI, camNearZ, nearScSize, scPixelSize
                //     );
                //     GET_POLY_ON_LINE_INXTN(
                //         result, i*pixelVsSize + targetIndex, targetIndex, cnvtPolyV2, cnvtPolyV3, vvFaceDot, 
                //         viewVolumeVs, vvFaceVsI, roopFaceI, camNearZ, nearScSize, scPixelSize
                //     );

                //     CALC_VV_FACE_DOT(vvFaceDot, cnvtPolyV3, cnvtPolyV1, viewVolumeVs, vvFaceI[roopFaceI], viewVolumeNs, roopFaceI);
                //     GET_POLY_ON_LINE_INXTN(
                //         result, i*pixelVsSize + targetIndex, targetIndex, cnvtPolyV3, cnvtPolyV1, vvFaceDot, 
                //         viewVolumeVs, vvFaceVsI, roopFaceI, camNearZ, nearScSize, scPixelSize
                //     );
                // }
                

            }

        }

        


    }

}

void Render::rasterize(std::unordered_map<std::wstring, Object> sObj, Camera cam, LPDWORD buffer)
{
    std::vector<float> polyVs;
    std::vector<float> polyNs;
    int loopObjI = 0;
    for (auto obj : sObj)
    {
        loopObjI++;
        if (hObjInJudgeAry[loopObjI] == FALSE)
        {
            continue;
        }

        for (int i = 0; i < obj.second.poly.vId.size(); i++)
        {
            polyVs.push_back(obj.second.v.world[obj.second.poly.vId[i].n1].x / CALC_SCALE);
            polyVs.push_back(obj.second.v.world[obj.second.poly.vId[i].n1].y / CALC_SCALE);
            polyVs.push_back(obj.second.v.world[obj.second.poly.vId[i].n1].z / CALC_SCALE);

            polyVs.push_back(obj.second.v.world[obj.second.poly.vId[i].n2].x / CALC_SCALE);
            polyVs.push_back(obj.second.v.world[obj.second.poly.vId[i].n2].y / CALC_SCALE);
            polyVs.push_back(obj.second.v.world[obj.second.poly.vId[i].n2].z / CALC_SCALE);

            polyVs.push_back(obj.second.v.world[obj.second.poly.vId[i].n3].x / CALC_SCALE);
            polyVs.push_back(obj.second.v.world[obj.second.poly.vId[i].n3].y / CALC_SCALE);
            polyVs.push_back(obj.second.v.world[obj.second.poly.vId[i].n3].z / CALC_SCALE);

            polyNs.push_back(obj.second.v.normal[obj.second.poly.normalId[i].n1].x);
            polyNs.push_back(obj.second.v.normal[obj.second.poly.normalId[i].n1].y);
            polyNs.push_back(obj.second.v.normal[obj.second.poly.normalId[i].n1].z);
        }
    }

    float* dPolyVs;
    float* dPolyNs;
    hipMalloc((void**)&dPolyVs, sizeof(float)*polyVs.size());
    hipMalloc((void**)&dPolyNs, sizeof(float)*polyNs.size());
    hipMemcpy(dPolyVs, polyVs.data(), sizeof(float)*polyVs.size(), hipMemcpyHostToDevice);
    hipMemcpy(dPolyNs, polyNs.data(), sizeof(float)*polyNs.size(), hipMemcpyHostToDevice);


    float* dMtCamTransRot;
    hipMalloc((void**)&dMtCamTransRot, sizeof(float)*hMtCamTransRot.size());
    hipMemcpy(dMtCamTransRot, hMtCamTransRot.data(), sizeof(float)*hMtCamTransRot.size(), hipMemcpyHostToDevice);


    hMtCamRot[0] = cos(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.y));
    hMtCamRot[1] = cos(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * sin(RAD(-cam.rotAngle.x)) + -sin(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.x));
    hMtCamRot[2] = cos(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * cos(RAD(-cam.rotAngle.x)) + -sin(RAD(-cam.rotAngle.z)) * -sin(RAD(-cam.rotAngle.x));
    hMtCamRot[3] = 0;
    hMtCamRot[4] = sin(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.y));
    hMtCamRot[5] = sin(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * sin(RAD(-cam.rotAngle.x)) + cos(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.x));
    hMtCamRot[6] = sin(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * cos(RAD(-cam.rotAngle.x)) + cos(RAD(-cam.rotAngle.z)) * -sin(RAD(-cam.rotAngle.x));
    hMtCamRot[7] = 0;
    hMtCamRot[8] = -sin(RAD(-cam.rotAngle.y));
    hMtCamRot[9] = cos(RAD(-cam.rotAngle.y)) * sin(RAD(-cam.rotAngle.x));
    hMtCamRot[10] = cos(RAD(-cam.rotAngle.y)) * cos(RAD(-cam.rotAngle.x));
    hMtCamRot[11] = 0;
    hMtCamRot[12] = 0;
    hMtCamRot[13] = 0;
    hMtCamRot[14] = 0;
    hMtCamRot[15] = 1;

    float* dMtCamRot;
    hipMalloc((void**)&dMtCamRot, sizeof(float)*hMtCamRot.size());
    hipMemcpy(dMtCamRot, hMtCamRot.data(), sizeof(float)*hMtCamRot.size(), hipMemcpyHostToDevice);


    float* dCamViewAngleCos;
    hipMalloc((void**)&dCamViewAngleCos, sizeof(float)*hCamViewAngleCos.size());
    hipMemcpy(dCamViewAngleCos, hCamViewAngleCos.data(), sizeof(float)*hCamViewAngleCos.size(), hipMemcpyHostToDevice);


    std::vector<float> hViewVolumeVs;
    for (int i = 0; i < 8; i++){
        hViewVolumeVs.push_back(cam.viewVolume.v[i].x / CALC_SCALE);
        hViewVolumeVs.push_back(cam.viewVolume.v[i].y / CALC_SCALE);
        hViewVolumeVs.push_back(cam.viewVolume.v[i].z / CALC_SCALE);
    }

    float* dViewVolumeVs;
    hipMalloc((void**)&dViewVolumeVs, sizeof(float)*hViewVolumeVs.size());
    hipMemcpy(dViewVolumeVs, hViewVolumeVs.data(), sizeof(float)*hViewVolumeVs.size(), hipMemcpyHostToDevice);


    std::vector<float> hViewVolumeNs;
    for (int i = 0; i < 6; i++){
        hViewVolumeNs.push_back(cam.viewVolume.face.normal[i].x / CALC_SCALE);
        hViewVolumeNs.push_back(cam.viewVolume.face.normal[i].y / CALC_SCALE);
        hViewVolumeNs.push_back(cam.viewVolume.face.normal[i].z / CALC_SCALE);
    }

    float* dViewVolumeNs;
    hipMalloc((void**)&dViewVolumeNs, sizeof(float)*hViewVolumeNs.size());
    hipMemcpy(dViewVolumeNs, hViewVolumeNs.data(), sizeof(float)*hViewVolumeNs.size(), hipMemcpyHostToDevice);


    std::vector<float> hNearScSize;
    hNearScSize.push_back(cam.nearScrSize.x / CALC_SCALE);
    hNearScSize.push_back(cam.nearScrSize.y / CALC_SCALE);

    float* dNearScSize;
    hipMalloc((void**)&dNearScSize, sizeof(float)*hNearScSize.size());
    hipMemcpy(dNearScSize, hNearScSize.data(), sizeof(float)*hNearScSize.size(), hipMemcpyHostToDevice);

    
    std::vector<float> hScPixelSize;
    hScPixelSize.push_back(cam.scPixelSize.x);
    hScPixelSize.push_back(cam.scPixelSize.y);

    float* dScPixelSize;
    hipMalloc((void**)&dScPixelSize, sizeof(float)*hScPixelSize.size());
    hipMemcpy(dScPixelSize, hScPixelSize.data(), sizeof(float)*hScPixelSize.size(), hipMemcpyHostToDevice);

    int resultSize = (polyNs.size() / 3) * (12 * 3 * 3 + 3*3 + 3*3);
    float* hResult = new float[resultSize];
    float* dResult;
    hipMalloc((void**)&dResult, sizeof(float)*resultSize);

    int polyAmount = polyNs.size() / 3;

    float camFarZ = static_cast<float>(cam.farZ / CALC_SCALE);
    float camNearZ = static_cast<float>(cam.nearZ / CALC_SCALE);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int dataSize = polyNs.size() / 3;
    int desiredThreadsPerBlock = 256;

    int blocks = (dataSize + desiredThreadsPerBlock - 1) / desiredThreadsPerBlock;
    int threadsPerBlock = std::min(desiredThreadsPerBlock, deviceProp.maxThreadsPerBlock);

    dim3 dimBlock(threadsPerBlock);
    dim3 dimGrid(blocks);
    
    glpaGpuRender<<<dimGrid, dimBlock>>>
    (
        dPolyVs, dPolyNs, polyAmount, 
        dMtCamTransRot, dMtCamRot, camFarZ, camNearZ,
        dCamViewAngleCos, dViewVolumeVs, dViewVolumeNs, dNearScSize, dScPixelSize, dResult
    );
    hipDeviceSynchronize();
    hipMemcpy(hResult, dResult, sizeof(float)*resultSize , hipMemcpyDeviceToHost);

    hipError_t error = hipGetLastError();
    if (error != 0)
    {
        throw std::runtime_error(ERROR_VECTOR_CUDA_ERROR);
    }

    delete[] hObjInJudgeAry;
    delete[] hResult;

    hipFree(dPolyVs);
    hipFree(dPolyNs);
    hipFree(dMtCamTransRot);
    hipFree(dMtCamRot);
    hipFree(dCamViewAngleCos);
    hipFree(dViewVolumeVs);
    hipFree(dViewVolumeNs);
    hipFree(dNearScSize);
    hipFree(dScPixelSize);
    hipFree(dResult);


}
