#include "hip/hip_runtime.h"
#include "render.cuh"

__global__ void glpaGpuPreparePoly(
    int objSize,
    float* objWVs,
    float* mtCamTransRot
){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < objSize)
    {
        float vec3d[3] = {objWVs[i*8*3 + AX], objWVs[i*8*3 + AY], objWVs[i*8*3 + AZ]};

        float camObjVs[3] = {
            vec3d[AX] * mtCamTransRot[0] + vec3d[AY] * mtCamTransRot[1] + vec3d[AZ] * mtCamTransRot[2] + 1 * mtCamTransRot[3],
            vec3d[AX] * mtCamTransRot[4] + vec3d[AY] * mtCamTransRot[5] + vec3d[AZ] * mtCamTransRot[6] + 1 * mtCamTransRot[7],
            vec3d[AX] * mtCamTransRot[8] + vec3d[AY] * mtCamTransRot[9] + vec3d[AZ] * mtCamTransRot[10] + 1 * mtCamTransRot[11]
        };
    }
}

void Render::prepareObjs(std::unordered_map<std::wstring, Object> sObj, Camera cam)
{
    int sObjSize = sObj.size();
    int objWvsSize = sObjSize*8*3;

    float* objWvs = new float[objWvsSize];

    int roopObj = 0;
    for (auto obj : sObj){
        for (int i = 0; i < 8; i++){
            objWvs[roopObj*8*3 + i*3] = obj.second.range.wVertex[i].x / CALC_SCALE;
            objWvs[roopObj*8*3 + i*3 + 1] = obj.second.range.wVertex[i].y / CALC_SCALE;
            objWvs[roopObj*8*3 + i*3 + 2] = obj.second.range.wVertex[i].z / CALC_SCALE;
        }
        roopObj += 1;
    }

    float* mtCamTransRot = new float[16];
    mtCamTransRot[0] = cos(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.y));
    mtCamTransRot[1] = cos(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * sin(RAD(-cam.rotAngle.x)) + -sin(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.x));
    mtCamTransRot[2] = cos(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * cos(RAD(-cam.rotAngle.x)) + -sin(RAD(-cam.rotAngle.z)) * -sin(RAD(-cam.rotAngle.x));
    mtCamTransRot[3] = -cam.wPos.x / CALC_SCALE;
    mtCamTransRot[4] = sin(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.y));
    mtCamTransRot[5] = sin(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * sin(RAD(-cam.rotAngle.x)) + cos(RAD(-cam.rotAngle.z)) * cos(RAD(-cam.rotAngle.x));
    mtCamTransRot[6] = sin(RAD(-cam.rotAngle.z)) * sin(RAD(-cam.rotAngle.y)) * cos(RAD(-cam.rotAngle.x)) + cos(RAD(-cam.rotAngle.z)) * -sin(RAD(-cam.rotAngle.x));
    mtCamTransRot[7] = -cam.wPos.y / CALC_SCALE;
    mtCamTransRot[8] = -sin(RAD(-cam.rotAngle.y));
    mtCamTransRot[9] = cos(RAD(-cam.rotAngle.y)) * sin(RAD(-cam.rotAngle.x));
    mtCamTransRot[10] = cos(RAD(-cam.rotAngle.y)) * cos(RAD(-cam.rotAngle.x));
    mtCamTransRot[11] = -cam.wPos.z / CALC_SCALE;
    mtCamTransRot[12] = 0;
    mtCamTransRot[13] = 0;
    mtCamTransRot[14] = 0;
    mtCamTransRot[15] = 1;


    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    int dataSizeY = sObj.size();
    int dataSizeX = 4; // 4 because there are two opposite v's on each of the xz plane and yz plane.

    int desiredThreadsPerBlockX = 16;
    int desiredThreadsPerBlockY = 16;

    int blocksX = (dataSizeX + desiredThreadsPerBlockX - 1) / desiredThreadsPerBlockX;
    int blocksY = (dataSizeY + desiredThreadsPerBlockY - 1) / desiredThreadsPerBlockY;

    int threadsPerBlockX = min(desiredThreadsPerBlockX, deviceProp.maxThreadsDim[0]);
    int threadsPerBlockY = min(desiredThreadsPerBlockY, deviceProp.maxThreadsDim[1]);

    dim3 dimBlock(threadsPerBlockX, threadsPerBlockY);
    dim3 dimGrid(blocksX, blocksY);

    glpaGpuPreparePoly<<<dimGrid, dimBlock>>>(

    );
}

void Render::render(std::unordered_map<std::wstring, Object> sObj, Camera cam, LPDWORD buffer){
    
}
