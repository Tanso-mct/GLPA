#include "hip/hip_runtime.h"
#include "matrix.cuh"


__global__ void glpaGpu4x4_4x1sMtProduct(double *mt4x4, double *mt4x1s, double *resultMt, int mt4x1sSize){
    // Decide which (i,j) you are in charge of based on your back number
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < mt4x1sSize && j < 3){
        resultMt[i*3 + j] 
        = mt4x1s[i*3 + 0] * mt4x4[j*4 + 0]
        + mt4x1s[i*3 + 1] * mt4x4[j*4 + 1]
        + mt4x1s[i*3 + 2] * mt4x4[j*4 + 2]
        + 1 * mt4x4[j*4 + 3];
    }
}


std::vector<Vec3d> Matrix::transRotConvert(Vec3d trans, Vec3d rot, std::vector<Vec3d> sourceVecs){
    hLeftMt = (double*)malloc(sizeof(double)*4*4);
    hRightMt = (double*)malloc(sizeof(double)*sourceVecs.size()*3);
    hResultMt = (double*)malloc(sizeof(double)*sourceVecs.size()*3);

    hLeftMt[0] = cos(RAD(rot.z)) * cos(RAD(rot.y));
    hLeftMt[1] = cos(RAD(rot.z)) * sin(RAD(rot.y)) * sin(RAD(rot.x)) * -sin(RAD(rot.z)) * cos(RAD(rot.x));
    hLeftMt[2] = cos(RAD(rot.z)) * sin(RAD(rot.y)) * cos(RAD(rot.x)) * -sin(RAD(rot.z)) * -sin(RAD(rot.x));
    hLeftMt[3] = cos(RAD(rot.z)) * cos(RAD(rot.y)) * trans.x * sin(RAD(rot.y)) * sin(RAD(rot.x)) * trans.y * cos(RAD(rot.x)) * trans.z * -sin(RAD(rot.z)) * cos(RAD(rot.x)) * trans.y * -sin(RAD(rot.x)) * trans.z;
    hLeftMt[4] = sin(RAD(rot.z)) * cos(RAD(rot.y));
    hLeftMt[5] = sin(RAD(rot.z)) * sin(RAD(rot.y)) * sin(RAD(rot.x)) * cos(RAD(rot.z)) * cos(RAD(rot.x));
    hLeftMt[6] = sin(RAD(rot.z)) * sin(RAD(rot.y)) * cos(RAD(rot.x)) * cos(RAD(rot.z)) * -sin(RAD(rot.x));
    hLeftMt[7] = sin(RAD(rot.z)) * cos(RAD(rot.y)) * trans.x * sin(RAD(rot.y)) * sin(RAD(rot.x)) * trans.y * cos(RAD(rot.x)) * trans.z * cos(RAD(rot.z)) * cos(RAD(rot.x)) * trans.y * -sin(RAD(rot.x)) * trans.z;
    hLeftMt[8] = -sin(RAD(rot.y));
    hLeftMt[9] = cos(RAD(rot.y)) * sin(RAD(rot.x));
    hLeftMt[10] = cos(RAD(rot.y)) * cos(RAD(rot.x));
    hLeftMt[11] = -sin(RAD(rot.y)) * trans.x * cos(RAD(rot.y)) * sin(RAD(rot.x)) * trans.y * cos(RAD(rot.x)) * trans.z;
    hLeftMt[12] = 0;
    hLeftMt[13] = 0;
    hLeftMt[14] = 0;
    hLeftMt[15] = 1;

    memcpy(hRightMt, sourceVecs.data(), sizeof(double)*sourceVecs.size()*3);

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dLeftMt, sizeof(double)*4*4);
    hipMalloc((void**)&dRightMt, sizeof(double)*sourceVecs.size()*3);
    hipMalloc((void**)&dResultMt, sizeof(double)*sourceVecs.size()*3);

    // Copy host-side data to device-side memory
    hipMemcpy(dLeftMt, hLeftMt, sizeof(double)*4*4, hipMemcpyHostToDevice);
    hipMemcpy(dRightMt, hRightMt, sizeof(double)*sourceVecs.size()*3, hipMemcpyHostToDevice);

    // GPU kernel function calls
    dim3 dimBlock(32, 32); // Thread block size
    dim3 dimGrid((sourceVecs.size() + dimBlock.x - 1) 
    / dimBlock.x, (sourceVecs.size() + dimBlock.y - 1) / dimBlock.y); // Grid Size
    glpaGpu4x4_4x1sMtProduct<<<dimGrid, dimBlock>>>
    (dLeftMt, dRightMt, dResultMt, sourceVecs.size());

    // Copy results from device memory to host memory
    hipMemcpy(hResultMt, dResultMt, sizeof(double)*sourceVecs.size()*3, hipMemcpyDeviceToHost);
    
    std::vector<Vec3d> rtCalcVec(sourceVecs.size());


    return std::vector<Vec3d>();
}

