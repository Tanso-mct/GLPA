#include "hip/hip_runtime.h"
#include "cgmath.cuh"

__global__ void gpuVecAddition
(
    double* sourceV, 
    double* calcV, 
    double* resultV, 
    int size // Number of array columns
)
{
    // Decide which (i,j) you are in charge of based on your back number
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size && j < VECTOR3)
    {
        resultV[i*VECTOR3 + j] = sourceV[i*VECTOR3 + j] + calcV[j];
    }
}

__global__ void gpuVecDotProduct
(
    double* sourceV, 
    double* calcV, 
    double* resultV, 
    int size
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        resultV[i]
        = sourceV[i*VECTOR3 + VX] * calcV[i*VECTOR3 + VX] / 
        sqrt(calcV[i*VECTOR3 + VX]*calcV[i*VECTOR3 + VX] + 
        calcV[i*VECTOR3 + VY]*calcV[i*VECTOR3 + VY] + calcV[i*VECTOR3 + VZ]*calcV[i*VECTOR3 + VZ])

        + sourceV[i*VECTOR3 + VY] * calcV[i*VECTOR3 + VY] / 
        sqrt(calcV[i*VECTOR3 + VX]*calcV[i*VECTOR3 + VX] 
        + calcV[i*VECTOR3 + VY]*calcV[i*VECTOR3 + VY] + calcV[i*VECTOR3 + VZ]*calcV[i*VECTOR3 + VZ])

        + sourceV[i*VECTOR3 + VZ] * calcV[i*VECTOR3 + VZ] / 
        sqrt(calcV[i*VECTOR3 + VX]*calcV[i*VECTOR3 + VX] 
        + calcV[i*VECTOR3 + VY]*calcV[i*VECTOR3 + VY] + calcV[i*VECTOR3 + VZ]*calcV[i*VECTOR3 + VZ]);
    }
}

__global__ void gpuVecCrossProduct
(
    double* sourceV, 
    double* calcV, 
    double* resultV, 
    int size
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        resultV[i*VECTOR3 + VX]
        = sourceV[i*VECTOR3 + VY] * calcV[i*VECTOR3 + VZ] - sourceV[i*VECTOR3 + VZ] * calcV[i*VECTOR3 + VY];

        resultV[i*VECTOR3 + VY]
        = sourceV[i*VECTOR3 + VZ] * calcV[i*VECTOR3 + VX] - sourceV[i*VECTOR3 + VX] * calcV[i*VECTOR3 + VZ];

        resultV[i*VECTOR3 + VZ]
        = sourceV[i*VECTOR3 + VX] * calcV[i*VECTOR3 + VY] - sourceV[i*VECTOR3 + VY] * calcV[i*VECTOR3 + VX];
    }
}

void VECTOR::pushVec3d
(
    double pushX,
    double pushY,
    double pushZ,
    std::vector<VECTOR3D>* inputVevotr3d
)
{
    VECTOR3D pushVec{pushX, pushY, pushZ};
    inputVevotr3d->push_back(pushVec);
}

void VECTOR::inputVec3d
(
    double inputX, 
    double inputY, 
    double inputZ, 
    int arrayNumInput, 
    std::vector<VECTOR3D>* inputVevotr3d
)
{
    (*inputVevotr3d)[arrayNumInput].x = inputX;
    (*inputVevotr3d)[arrayNumInput].y = inputY;
    (*inputVevotr3d)[arrayNumInput].z = inputZ;
}

void VECTOR::posTrans(std::vector<VECTOR3D> sourceVec, VECTOR3D calcVec)
{
    // Allocate memory for each vector size
    hSouceVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());
    hCalcVec = (double*)malloc(sizeof(double)*VECTOR3);
    hResultVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());

    memcpy(hSouceVec, sourceVec.data(), sizeof(double)*VECTOR3*sourceVec.size());

    hCalcVec[VX] = calcVec.x;
    hCalcVec[VY] = calcVec.y;
    hCalcVec[VZ] = calcVec.z;

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dSouceVec, sizeof(double)*VECTOR3*sourceVec.size());
    hipMalloc((void**)&dCalcVec, sizeof(double)*VECTOR3);
    hipMalloc((void**)&dResultVec, sizeof(double)*VECTOR3*sourceVec.size());

    // Copy host-side data to device-side memory
    hipMemcpy(dSouceVec, hSouceVec, sizeof(double)*VECTOR3*sourceVec.size(), hipMemcpyHostToDevice);
    hipMemcpy(dCalcVec, hCalcVec, sizeof(double)*VECTOR3, hipMemcpyHostToDevice);
    
    // GPU kernel function calls
    dim3 dimBlock(32, 32); // Thread block size
    dim3 dimGrid
    (
        (sourceVec.size()*VECTOR3 + dimBlock.x - 1) / dimBlock.x, 
        (sourceVec.size()*VECTOR3 + dimBlock.y - 1) / dimBlock.y
    ); // Grid Size
    gpuVecAddition<<<dimGrid, dimBlock>>>
    (dSouceVec, dCalcVec, dResultVec, sourceVec.size());

    // Copy results from device memory to host memory
    hipMemcpy(hResultVec, dResultVec, sizeof(double)*3*sourceVec.size(), hipMemcpyDeviceToHost);

    // Assign the result to a Vector member variable
    resultVector3D.resize(sourceVec.size());
    for (int i = 0; i < sourceVec.size(); ++i)
    {
        inputVec3d
        (
            hResultVec[i*VECTOR3 + VX], 
            hResultVec[i*VECTOR3 + VY], 
            hResultVec[i*VECTOR3 + VZ], 
            i,
            &resultVector3D
        );
    }

    // Release all memory allocated by malloc
    free(hSouceVec);
    free(hCalcVec);
    free(hResultVec);

    hipFree(dSouceVec);
    hipFree(dCalcVec);
    hipFree(dResultVec);
}

void VECTOR::dotProduct(std::vector<VECTOR3D> sourceVec, std::vector<VECTOR3D> calcVec)
{
    if (sourceVec.size() != calcVec.size())
    {
        OutputDebugStringA("Vector function{dotProduct} ERROR\n");
        OutputDebugStringA("souceVec and calcVec array sizes are different\n");
        return;
    }
    // Allocate memory for each vector size
    hSouceVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());
    hCalcVec = (double*)malloc(sizeof(double)*VECTOR3*calcVec.size());
    hResultVec = (double*)malloc(sizeof(double)*sourceVec.size());

    memcpy(hSouceVec, sourceVec.data(), sizeof(double)*3*sourceVec.size());

    for (int i = 0; i < calcVec.size(); ++i)
    {
        hCalcVec[i*VECTOR3 + 0] = calcVec[i].x;
        hCalcVec[i*VECTOR3 + 1] = calcVec[i].y;
        hCalcVec[i*VECTOR3 + 2] = calcVec[i].z;
    }

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dSouceVec, sizeof(double)*VECTOR3*sourceVec.size());
    hipMalloc((void**)&dCalcVec, sizeof(double)*VECTOR3*calcVec.size());
    hipMalloc((void**)&dResultVec, sizeof(double)*sourceVec.size());

    // Copy host-side data to device-side memory
    hipMemcpy(dSouceVec, hSouceVec, sizeof(double)*VECTOR3*sourceVec.size(), hipMemcpyHostToDevice);
    hipMemcpy(dCalcVec, hCalcVec, sizeof(double)*VECTOR3*calcVec.size(), hipMemcpyHostToDevice);
    
    // GPU kernel function calls
    int blockSize = 1024;
    int numBlocks = (sourceVec.size() + blockSize - 1) / blockSize;
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
    gpuVecDotProduct<<<dimGrid, dimBlock>>>
    (dSouceVec, dCalcVec, dResultVec, sourceVec.size());

    // Copy results from device memory to host memory
    hipMemcpy(hResultVec, dResultVec, sizeof(double)*sourceVec.size(), hipMemcpyDeviceToHost);


    // Assign the result to a Vector member variable
    resultVector.resize(sourceVec.size());
    for (int i = 0; i < sourceVec.size(); ++i)
    {
        resultVector[i] = hResultVec[i];
    }

    // Release all memory allocated by malloc
    free(hSouceVec);
    free(hCalcVec);
    free(hResultVec);

    hipFree(dSouceVec);
    hipFree(dCalcVec);
    hipFree(dResultVec);

}

void VECTOR::crossProduct(std::vector<VECTOR3D> sourceVec, std::vector<VECTOR3D> calcVec)
{
    if (sourceVec.size() != calcVec.size())
    {
        OutputDebugStringA("Vector function{crossProduct} ERROR\n");
        OutputDebugStringA("souceVec and calcVec array sizes are different\n");
        return;
    }

    // Allocate memory for each vector size
    hSouceVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());
    hCalcVec = (double*)malloc(sizeof(double)*VECTOR3*calcVec.size());
    hResultVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());

    memcpy(hSouceVec, sourceVec.data(), sizeof(double)*3*sourceVec.size());
    memcpy(hCalcVec, calcVec.data(), sizeof(double)*3*calcVec.size());

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dSouceVec, sizeof(double)*VECTOR3*sourceVec.size());
    hipMalloc((void**)&dCalcVec, sizeof(double)*VECTOR3*calcVec.size());
    hipMalloc((void**)&dResultVec, sizeof(double)*VECTOR3*sourceVec.size());

    // Copy host-side data to device-side memory
    hipMemcpy(dSouceVec, hSouceVec, sizeof(double)*VECTOR3*sourceVec.size(), hipMemcpyHostToDevice);
    hipMemcpy(dCalcVec, hCalcVec, sizeof(double)*VECTOR3*calcVec.size(), hipMemcpyHostToDevice);
    
    // GPU kernel function calls
    int blockSize = 1024;
    int numBlocks = (sourceVec.size()*VECTOR3 + blockSize - 1) / blockSize;
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
    (
        (sourceVec.size()*VECTOR3 + dimBlock.x - 1) / dimBlock.x, 
        (sourceVec.size()*VECTOR3 + dimBlock.y - 1) / dimBlock.y
    ); // Grid Size
    gpuVecCrossProduct<<<dimGrid, dimBlock>>>
    (dSouceVec, dCalcVec, dResultVec, sourceVec.size());

    // Copy results from device memory to host memory
    hipMemcpy(hResultVec, dResultVec, sizeof(double)*VECTOR3*sourceVec.size(), hipMemcpyDeviceToHost);

    // Assign the result to a Vector member variable
    resultVector3D.resize(sourceVec.size());
    for (int i = 0; i < sourceVec.size(); ++i)
    {
        inputVec3d
        (
            hResultVec[i*VECTOR3 + VX], 
            hResultVec[i*VECTOR3 + VY], 
            hResultVec[i*VECTOR3 + VZ], 
            i,
            &resultVector3D
        );
    }

    // Release all memory allocated by malloc
    free(hSouceVec);
    free(hCalcVec);
    free(hResultVec);

    hipFree(dSouceVec);
    hipFree(dCalcVec);
    hipFree(dResultVec);
}



void MATRIX::input3xMatrix
(
    std::vector<VECTOR3D> *inputMatrix, 
    double a11, double a12, double a13, 
    double a21, double a22, double a23, 
    double a31, double a32, double a33
)
{
    (*inputMatrix)[C1].x = a11;
    (*inputMatrix)[C1].y = a21;
    (*inputMatrix)[C1].z = a31;

    (*inputMatrix)[C2].x = a12;
    (*inputMatrix)[C2].y = a22;
    (*inputMatrix)[C2].z = a32;

    (*inputMatrix)[C3].x = a13;
    (*inputMatrix)[C3].y = a23;
    (*inputMatrix)[C3].z = a33;
}

__global__ void gpuCalc3xMatrixProduct
(
    double* sourceMatrices, 
    double* calcMatrices, 
    double* resultMatrices, 
    int size
)
{
    // Decide which (i,j) you are in charge of based on your back number
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size && j < MATRIX3RAW)
    {
        resultMatrices[MATRIX3RAW*i + j] 
        = sourceMatrices[i*MATRIX3RAW + C1] * calcMatrices[j + MATRIX3RAW*R1] 
        + sourceMatrices[i*MATRIX3RAW + C2] * calcMatrices[j + MATRIX3RAW*R2]
        + sourceMatrices[i*MATRIX3RAW + C3] * calcMatrices[j + MATRIX3RAW*R3];
    }

}

void MATRIX::calcMatrix3xProduct()
{
    // Allocate memory for each matrix size
    hSourceMatrices = (double*)malloc(sizeof(double)*MATRIX3RAW*sourceMatrices.size());
    hCalcMatrices = (double*)malloc(sizeof(double)*MATRIX3RAW*calcMatrices3x.size());
    hResultMatrices = (double*)malloc(sizeof(double)*MATRIX3RAW*sourceMatrices.size());

    // Copy member variable
    memcpy(hSourceMatrices, sourceMatrices.data(), sizeof(double)*MATRIX3RAW*sourceMatrices.size());

    for (int i = 0; i < calcMatrices3x.size(); ++i)
    {
        hCalcMatrices[i*MATRIX3RAW+C1] = calcMatrices3x[i].x;
        hCalcMatrices[i*MATRIX3RAW+C2] = calcMatrices3x[i].y;
        hCalcMatrices[i*MATRIX3RAW+C3] = calcMatrices3x[i].z;
    }

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dSourceMatrices, sizeof(double)*MATRIX3RAW*sourceMatrices.size());
    hipMalloc((void**)&dCalcMatrices, sizeof(double)*MATRIX3RAW*calcMatrices3x.size());
    hipMalloc((void**)&dResultMatrices, sizeof(double)*MATRIX3RAW*sourceMatrices.size());

    // Copy host-side data to device-side memory
    hipMemcpy(dSourceMatrices, hSourceMatrices, sizeof(double)*MATRIX3RAW*sourceMatrices.size(), hipMemcpyHostToDevice);
    hipMemcpy(dCalcMatrices, hCalcMatrices, sizeof(double)*MATRIX3RAW*calcMatrices3x.size(), hipMemcpyHostToDevice);
    // hipMemcpy(dResultMatrices, hResultMatrices, sizeof(double)*matrixRaw*sourceMatrices.size(), hipMemcpyHostToDevice);

    // GPU kernel function calls
    dim3 dimBlock(32, 32); // Thread block size
    dim3 dimGrid((sourceMatrices.size() + dimBlock.x - 1) 
    / dimBlock.x, (sourceMatrices.size() + dimBlock.y - 1) / dimBlock.y); // Grid Size
    gpuCalc3xMatrixProduct<<<dimGrid, dimBlock>>>
    (dSourceMatrices, dCalcMatrices, dResultMatrices, sourceMatrices.size());

    // Copy results from device memory to host memory
    hipMemcpy(hResultMatrices, dResultMatrices, sizeof(double)*MATRIX3RAW*sourceMatrices.size(), hipMemcpyDeviceToHost);
    
    // Assign the result to a Vector member variable
    resultMatrices.resize(sourceMatrices.size());
    for (int i = 0; i < sourceMatrices.size(); ++i)
    {
        vec.inputVec3d
        (
            hResultMatrices[i*MATRIX3RAW+C1], 
            hResultMatrices[i*MATRIX3RAW+C2], 
            hResultMatrices[i*MATRIX3RAW+C3], 
            i,
            &resultMatrices
        );
    }

    // Release all memory allocated by malloc
    free(hSourceMatrices);
    free(hCalcMatrices);
    free(hResultMatrices);

    hipFree(dSourceMatrices);
    hipFree(dCalcMatrices);
    hipFree(dResultMatrices);
}

void MATRIX::rotTrans(std::vector<VECTOR3D> sourceCoordinates, VECTOR3D rotAngle)
{
    sourceMatrices.resize(sourceCoordinates.size());
    sourceMatrices = sourceCoordinates;

    double calcRotAngle;

    calcRotAngle = rotAngle.x;
    input3xMatrix
    (
        &calcMatrices3x,
        1,    0,                               0, 
        0,    cos(calcRotAngle * PI / 180),    -sin(calcRotAngle * PI / 180),
        0,    sin(calcRotAngle * PI / 180),    cos(calcRotAngle * PI / 180)
    );
    calcMatrix3xProduct();
    sourceMatrices = resultMatrices;

    calcRotAngle = rotAngle.y;
    input3xMatrix
    (
        &calcMatrices3x,
        cos(calcRotAngle * PI / 180),     0,     sin(calcRotAngle * PI / 180), 
        0,                                1,    0,
        -sin(calcRotAngle * PI / 180),    0,     cos(calcRotAngle * PI / 180)
    );
    calcMatrix3xProduct();
    sourceMatrices = resultMatrices;
        
    calcRotAngle = rotAngle.z;
    input3xMatrix
    (
        &calcMatrices3x,
        cos(calcRotAngle * PI / 180),     -sin(calcRotAngle * PI / 180),   0, 
        sin(calcRotAngle * PI / 180),     cos(calcRotAngle * PI / 180),    0,
        0,                                0,                               1
    );
    calcMatrix3xProduct();
    sourceMatrices = resultMatrices;

}

void MATRIX::scaleTrans(std::vector<VECTOR3D> sourceCoordinates, VECTOR3D scalingRate)
{
    sourceMatrices.resize(sourceCoordinates.size());
    sourceMatrices = sourceCoordinates;

    input3xMatrix
    (
        &calcMatrices3x,
        scalingRate.x,  0,              0,
        0,              scalingRate.y,  0,
        0,              0,              scalingRate.z
    );

    calcMatrix3xProduct();
}

__global__ void gpuGetLinePlaneI
(
    double* lineVA, // x1, y1, z1
    double* lineVB, // l, m, n
    double* planeV, // x0, y0, z0
    double* planeN, // p, q, r
    double* lpI,
    int lineAmout,
    int planeAmout
)
{
    // Decide which (i,j) you are in charge of based on your back number
    int i = blockIdx.z * blockDim.z + threadIdx.z;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < lineAmout && j < planeAmout && k < VECTOR3)
    {
        lpI[i*planeAmout + j*VECTOR3 + k] = 
        lineVA[i*VECTOR3 + k] + 
        ((planeN[j*VECTOR3 + PX] * (-lineVA[i*VECTOR3 + X1] + planeV[j*VECTOR3 + X0])
        + planeN[j*VECTOR3 + QY] * (-lineVA[i*VECTOR3 + Y1] + planeV[j*VECTOR3 + Y0])
        + planeN[j*VECTOR3 + RZ] * (-lineVA[i*VECTOR3 + Z1] + planeV[j*VECTOR3 + Z0]))
        / (lineVB[i*VECTOR3 + X0] * planeN[j*VECTOR3 + X0]
        + lineVB[i*VECTOR3 + Y0] * planeN[j*VECTOR3 + Y0]
        + lineVB[i*VECTOR3 + Z0] * planeN[j*VECTOR3 + Z0]))
        * lineVB[i*VECTOR3 + k];
    }
}

void EQUATION::getLinePlaneI
(
    std::vector<VECTOR3D> lineVA,
    std::vector<VECTOR3D> lineVB,
    std::vector<VECTOR3D> planeV,
    std::vector<VECTOR3D> planeN
)
{
    int lineAmout = lineVA.size();
    int planeAmout = planeN.size();
    // Allocate memory for each matrix size
    hLineVertexA = (double*)malloc(sizeof(double)*VECTOR3*lineAmout);
    hLineVertexB = (double*)malloc(sizeof(double)*VECTOR3*lineAmout);
    hPlaneVertex = (double*)malloc(sizeof(double)*VECTOR3*planeAmout);
    hPlaneNormal = (double*)malloc(sizeof(double)*VECTOR3*planeAmout);
    hLinePlaneI = (double*)malloc(sizeof(double)*VECTOR3*lineAmout*planeAmout);

    // Copy member variable
    memcpy(hLineVertexA, lineVA.data(), sizeof(double)*VECTOR3*lineAmout);
    memcpy(hLineVertexB, lineVB.data(), sizeof(double)*VECTOR3*lineAmout);
    memcpy(hPlaneVertex, planeV.data(), sizeof(double)*VECTOR3*planeAmout);
    memcpy(hPlaneNormal, planeN.data(), sizeof(double)*VECTOR3*planeAmout);

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dLineVertexA, sizeof(double)*VECTOR3*lineAmout);
    hipMalloc((void**)&dLineVertexB, sizeof(double)*VECTOR3*lineAmout);
    hipMalloc((void**)&dPlaneVertex, sizeof(double)*VECTOR3*planeAmout);
    hipMalloc((void**)&dPlaneNormal, sizeof(double)*VECTOR3*planeAmout);
    hipMalloc((void**)&dLinePlaneI, sizeof(double)*VECTOR3*lineAmout*planeAmout);

    // Copy host-side data to device-side memory
    hipMemcpy(dLineVertexA, hLineVertexA, sizeof(double)*VECTOR3*lineAmout, hipMemcpyHostToDevice);
    hipMemcpy(dLineVertexB, hLineVertexB, sizeof(double)*VECTOR3*lineAmout, hipMemcpyHostToDevice);
    hipMemcpy(dPlaneVertex, hPlaneVertex, sizeof(double)*VECTOR3*planeAmout, hipMemcpyHostToDevice);
    hipMemcpy(dPlaneNormal, hPlaneNormal, sizeof(double)*VECTOR3*planeAmout, hipMemcpyHostToDevice);

    // GPU kernel function calls
    int blockSize = 32;
    dim3 dimBlock(blockSize, blockSize, blockSize);
    dim3 dimGrid
    (
        (VECTOR3 + blockSize - 1) / blockSize, 
        (planeN.size() + blockSize - 1) / blockSize, 
        (lineVA.size() + blockSize - 1) / blockSize
    );
    gpuGetLinePlaneI<<<dimGrid, dimBlock>>>
    (dLineVertexA, dLineVertexB, dPlaneVertex, dPlaneNormal, dLinePlaneI, lineVA.size(), planeN.size());

    // Copy results from device memory to host memory
    hipMemcpy(hLinePlaneI, dLinePlaneI, sizeof(double)*VECTOR3*lineAmout*planeAmout, hipMemcpyDeviceToHost);
    
    // Assign the result to a Vector member variable
    linePlaneI.resize(lineAmout*planeAmout);
    for (int i = 0; i < lineAmout*planeAmout; ++i)
    {
        vec.inputVec3d
        (
            hLinePlaneI[i*VECTOR3+X0], 
            hLinePlaneI[i*VECTOR3+Y0], 
            hLinePlaneI[i*VECTOR3+Z0], 
            i,
            &linePlaneI
        );
    }

    // Release all memory allocated by malloc
    free(hLineVertexA);
    free(hLineVertexB);
    free(hPlaneVertex);
    free(hPlaneNormal);
    free(hLinePlaneI);

    hipFree(dLineVertexA);
    hipFree(dLineVertexB);
    hipFree(dPlaneVertex);
    hipFree(dPlaneNormal);
    hipFree(dLinePlaneI);
}

