#include "hip/hip_runtime.h"
#include "cgmath.cuh"

__global__ void gpuVecAddition
(
    double* sourceV, 
    double* calcV, 
    double* resultV, 
    int size // Number of array columns
)
{
    // Decide which (i,j) you are in charge of based on your back number
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size && j < VECTOR3)
    {
        resultV[i*VECTOR3 + j] = sourceV[i*VECTOR3 + j] + calcV[j];
    }
}

__global__ void gpuVecDotProduct
(
    double* sourceV, 
    double* calcV, 
    double* resultV, 
    int size
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        resultV[i]
        = sourceV[i*VECTOR3 + VX] * calcV[i*VECTOR3 + VX] / 
        sqrt(calcV[i*VECTOR3 + VX]*calcV[i*VECTOR3 + VX] + 
        calcV[i*VECTOR3 + VY]*calcV[i*VECTOR3 + VY] + calcV[i*VECTOR3 + VZ]*calcV[i*VECTOR3 + VZ])

        + sourceV[i*VECTOR3 + VY] * calcV[i*VECTOR3 + VY] / 
        sqrt(calcV[i*VECTOR3 + VX]*calcV[i*VECTOR3 + VX] 
        + calcV[i*VECTOR3 + VY]*calcV[i*VECTOR3 + VY] + calcV[i*VECTOR3 + VZ]*calcV[i*VECTOR3 + VZ])

        + sourceV[i*VECTOR3 + VZ] * calcV[i*VECTOR3 + VZ] / 
        sqrt(calcV[i*VECTOR3 + VX]*calcV[i*VECTOR3 + VX] 
        + calcV[i*VECTOR3 + VY]*calcV[i*VECTOR3 + VY] + calcV[i*VECTOR3 + VZ]*calcV[i*VECTOR3 + VZ]);
    }
}

__global__ void gpuVecCrossProduct
(
    double* sourceV, 
    double* calcV, 
    double* resultV, 
    int size
)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size)
    {
        resultV[i*VECTOR3 + VX]
        = sourceV[i*VECTOR3 + VY] * calcV[i*VECTOR3 + VZ] - sourceV[i*VECTOR3 + VZ] * calcV[i*VECTOR3 + VY];

        resultV[i*VECTOR3 + VY]
        = sourceV[i*VECTOR3 + VZ] * calcV[i*VECTOR3 + VX] - sourceV[i*VECTOR3 + VX] * calcV[i*VECTOR3 + VZ];

        resultV[i*VECTOR3 + VZ]
        = sourceV[i*VECTOR3 + VX] * calcV[i*VECTOR3 + VY] - sourceV[i*VECTOR3 + VY] * calcV[i*VECTOR3 + VX];
    }
}

void VECTOR::minusVec3d(VECTOR3D a, VECTOR3D b, VECTOR3D *result)
{
    (*result).x = b.x - a.x;
    (*result).y = b.y - a.y;
    (*result).z = b.z - a.z;
}

void VECTOR::decimalLimit(VECTOR3D *v)
{
    (*v).x = std::floor(v->x * DECIMAL_PLACES) / DECIMAL_PLACES;
    (*v).y = std::floor(v->y * DECIMAL_PLACES) / DECIMAL_PLACES;
    (*v).z = std::floor(v->z * DECIMAL_PLACES) / DECIMAL_PLACES;
}

void VECTOR::pushVec3d
(
    double pushX,
    double pushY,
    double pushZ,
    std::vector<VECTOR3D>* inputVevotr3d
)
{
    VECTOR3D pushVec{pushX, pushY, pushZ};
    inputVevotr3d->push_back(pushVec);
}

void VECTOR::inputVec3d
(
    double inputX, 
    double inputY, 
    double inputZ, 
    int arrayNumInput, 
    std::vector<VECTOR3D>* inputVevotr3d
)
{
    (*inputVevotr3d)[arrayNumInput].x = inputX;
    (*inputVevotr3d)[arrayNumInput].y = inputY;
    (*inputVevotr3d)[arrayNumInput].z = inputZ;
}

void VECTOR::posTrans(std::vector<VECTOR3D> sourceVec, VECTOR3D calcVec)
{
    // Allocate memory for each vector size
    hSouceVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());
    hCalcVec = (double*)malloc(sizeof(double)*VECTOR3);
    hResultVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());

    memcpy(hSouceVec, sourceVec.data(), sizeof(double)*VECTOR3*sourceVec.size());

    hCalcVec[VX] = calcVec.x;
    hCalcVec[VY] = calcVec.y;
    hCalcVec[VZ] = calcVec.z;

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dSouceVec, sizeof(double)*VECTOR3*sourceVec.size());
    hipMalloc((void**)&dCalcVec, sizeof(double)*VECTOR3);
    hipMalloc((void**)&dResultVec, sizeof(double)*VECTOR3*sourceVec.size());

    // Copy host-side data to device-side memory
    hipMemcpy(dSouceVec, hSouceVec, sizeof(double)*VECTOR3*sourceVec.size(), hipMemcpyHostToDevice);
    hipMemcpy(dCalcVec, hCalcVec, sizeof(double)*VECTOR3, hipMemcpyHostToDevice);
    
    // GPU kernel function calls
    dim3 dimBlock(32, 32); // Thread block size
    dim3 dimGrid
    (
        (sourceVec.size()*VECTOR3 + dimBlock.x - 1) / dimBlock.x, 
        (sourceVec.size()*VECTOR3 + dimBlock.y - 1) / dimBlock.y
    ); // Grid Size
    gpuVecAddition<<<dimGrid, dimBlock>>>
    (dSouceVec, dCalcVec, dResultVec, sourceVec.size());

    // Copy results from device memory to host memory
    hipMemcpy(hResultVec, dResultVec, sizeof(double)*3*sourceVec.size(), hipMemcpyDeviceToHost);

    // Assign the result to a Vector member variable
    resultVector3D.resize(sourceVec.size());
    for (int i = 0; i < sourceVec.size(); ++i)
    {
        inputVec3d
        (
            hResultVec[i*VECTOR3 + VX], 
            hResultVec[i*VECTOR3 + VY], 
            hResultVec[i*VECTOR3 + VZ], 
            i,
            &resultVector3D
        );
    }

    // Release all memory allocated by malloc
    free(hSouceVec);
    free(hCalcVec);
    free(hResultVec);

    hipFree(dSouceVec);
    hipFree(dCalcVec);
    hipFree(dResultVec);
}

void VECTOR::dotProduct(std::vector<VECTOR3D> sourceVec, std::vector<VECTOR3D> calcVec)
{
    if (sourceVec.size() != calcVec.size())
    {
        OutputDebugStringA("Vector function{dotProduct} ERROR\n");
        OutputDebugStringA("souceVec and calcVec array sizes are different\n");
        return;
    }
    // Allocate memory for each vector size
    hSouceVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());
    hCalcVec = (double*)malloc(sizeof(double)*VECTOR3*calcVec.size());
    hResultVec = (double*)malloc(sizeof(double)*sourceVec.size());

    memcpy(hSouceVec, sourceVec.data(), sizeof(double)*3*sourceVec.size());

    for (int i = 0; i < calcVec.size(); ++i)
    {
        hCalcVec[i*VECTOR3 + 0] = calcVec[i].x;
        hCalcVec[i*VECTOR3 + 1] = calcVec[i].y;
        hCalcVec[i*VECTOR3 + 2] = calcVec[i].z;
    }

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dSouceVec, sizeof(double)*VECTOR3*sourceVec.size());
    hipMalloc((void**)&dCalcVec, sizeof(double)*VECTOR3*calcVec.size());
    hipMalloc((void**)&dResultVec, sizeof(double)*sourceVec.size());

    // Copy host-side data to device-side memory
    hipMemcpy(dSouceVec, hSouceVec, sizeof(double)*VECTOR3*sourceVec.size(), hipMemcpyHostToDevice);
    hipMemcpy(dCalcVec, hCalcVec, sizeof(double)*VECTOR3*calcVec.size(), hipMemcpyHostToDevice);
    
    // GPU kernel function calls
    int blockSize = 1024;
    int numBlocks = (sourceVec.size() + blockSize - 1) / blockSize;
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
    gpuVecDotProduct<<<dimGrid, dimBlock>>>
    (dSouceVec, dCalcVec, dResultVec, sourceVec.size());

    // Copy results from device memory to host memory
    hipMemcpy(hResultVec, dResultVec, sizeof(double)*sourceVec.size(), hipMemcpyDeviceToHost);


    // Assign the result to a Vector member variable
    resultVector.resize(sourceVec.size());
    for (int i = 0; i < sourceVec.size(); ++i)
    {
        resultVector[i] = hResultVec[i];
    }

    // Release all memory allocated by malloc
    free(hSouceVec);
    free(hCalcVec);
    free(hResultVec);

    hipFree(dSouceVec);
    hipFree(dCalcVec);
    hipFree(dResultVec);

}

void VECTOR::crossProduct(std::vector<VECTOR3D> sourceVec, std::vector<VECTOR3D> calcVec)
{
    if (sourceVec.size() != calcVec.size())
    {
        OutputDebugStringA("Vector function{crossProduct} ERROR\n");
        OutputDebugStringA("souceVec and calcVec array sizes are different\n");
        return;
    }

    // Allocate memory for each vector size
    hSouceVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());
    hCalcVec = (double*)malloc(sizeof(double)*VECTOR3*calcVec.size());
    hResultVec = (double*)malloc(sizeof(double)*VECTOR3*sourceVec.size());

    memcpy(hSouceVec, sourceVec.data(), sizeof(double)*3*sourceVec.size());
    memcpy(hCalcVec, calcVec.data(), sizeof(double)*3*calcVec.size());

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dSouceVec, sizeof(double)*VECTOR3*sourceVec.size());
    hipMalloc((void**)&dCalcVec, sizeof(double)*VECTOR3*calcVec.size());
    hipMalloc((void**)&dResultVec, sizeof(double)*VECTOR3*sourceVec.size());

    // Copy host-side data to device-side memory
    hipMemcpy(dSouceVec, hSouceVec, sizeof(double)*VECTOR3*sourceVec.size(), hipMemcpyHostToDevice);
    hipMemcpy(dCalcVec, hCalcVec, sizeof(double)*VECTOR3*calcVec.size(), hipMemcpyHostToDevice);
    
    // GPU kernel function calls
    int blockSize = 1024;
    int numBlocks = (sourceVec.size()*VECTOR3 + blockSize - 1) / blockSize;
    dim3 dimBlock(blockSize, 1, 1);
    dim3 dimGrid(numBlocks, 1, 1);
    (
        (sourceVec.size()*VECTOR3 + dimBlock.x - 1) / dimBlock.x, 
        (sourceVec.size()*VECTOR3 + dimBlock.y - 1) / dimBlock.y
    ); // Grid Size
    gpuVecCrossProduct<<<dimGrid, dimBlock>>>
    (dSouceVec, dCalcVec, dResultVec, sourceVec.size());

    // Copy results from device memory to host memory
    hipMemcpy(hResultVec, dResultVec, sizeof(double)*VECTOR3*sourceVec.size(), hipMemcpyDeviceToHost);

    // Assign the result to a Vector member variable
    resultVector3D.resize(sourceVec.size());
    for (int i = 0; i < sourceVec.size(); ++i)
    {
        inputVec3d
        (
            hResultVec[i*VECTOR3 + VX], 
            hResultVec[i*VECTOR3 + VY], 
            hResultVec[i*VECTOR3 + VZ], 
            i,
            &resultVector3D
        );
    }

    // Release all memory allocated by malloc
    free(hSouceVec);
    free(hCalcVec);
    free(hResultVec);

    hipFree(dSouceVec);
    hipFree(dCalcVec);
    hipFree(dResultVec);
}



void MATRIX::input3xMatrix
(
    std::vector<VECTOR3D> *inputMatrix, 
    double a11, double a12, double a13, 
    double a21, double a22, double a23, 
    double a31, double a32, double a33
)
{
    (*inputMatrix)[C1].x = a11;
    (*inputMatrix)[C1].y = a21;
    (*inputMatrix)[C1].z = a31;

    (*inputMatrix)[C2].x = a12;
    (*inputMatrix)[C2].y = a22;
    (*inputMatrix)[C2].z = a32;

    (*inputMatrix)[C3].x = a13;
    (*inputMatrix)[C3].y = a23;
    (*inputMatrix)[C3].z = a33;
}

__global__ void gpuCalc3xMatrixProduct
(
    double* sourceMatrices, 
    double* calcMatrices, 
    double* resultMatrices, 
    int size
)
{
    // Decide which (i,j) you are in charge of based on your back number
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size && j < MATRIX3RAW)
    {
        resultMatrices[MATRIX3RAW*i + j] 
        = sourceMatrices[i*MATRIX3RAW + C1] * calcMatrices[j + MATRIX3RAW*R1] 
        + sourceMatrices[i*MATRIX3RAW + C2] * calcMatrices[j + MATRIX3RAW*R2]
        + sourceMatrices[i*MATRIX3RAW + C3] * calcMatrices[j + MATRIX3RAW*R3];
    }

}

void MATRIX::calcMatrix3xProduct()
{
    // Allocate memory for each matrix size
    hSourceMatrices = (double*)malloc(sizeof(double)*MATRIX3RAW*sourceMatrices.size());
    hCalcMatrices = (double*)malloc(sizeof(double)*MATRIX3RAW*calcMatrices3x.size());
    hResultMatrices = (double*)malloc(sizeof(double)*MATRIX3RAW*sourceMatrices.size());

    // Copy member variable
    memcpy(hSourceMatrices, sourceMatrices.data(), sizeof(double)*MATRIX3RAW*sourceMatrices.size());

    for (int i = 0; i < calcMatrices3x.size(); ++i)
    {
        hCalcMatrices[i*MATRIX3RAW+C1] = calcMatrices3x[i].x;
        hCalcMatrices[i*MATRIX3RAW+C2] = calcMatrices3x[i].y;
        hCalcMatrices[i*MATRIX3RAW+C3] = calcMatrices3x[i].z;
    }

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dSourceMatrices, sizeof(double)*MATRIX3RAW*sourceMatrices.size());
    hipMalloc((void**)&dCalcMatrices, sizeof(double)*MATRIX3RAW*calcMatrices3x.size());
    hipMalloc((void**)&dResultMatrices, sizeof(double)*MATRIX3RAW*sourceMatrices.size());

    // Copy host-side data to device-side memory
    hipMemcpy(dSourceMatrices, hSourceMatrices, sizeof(double)*MATRIX3RAW*sourceMatrices.size(), hipMemcpyHostToDevice);
    hipMemcpy(dCalcMatrices, hCalcMatrices, sizeof(double)*MATRIX3RAW*calcMatrices3x.size(), hipMemcpyHostToDevice);
    // hipMemcpy(dResultMatrices, hResultMatrices, sizeof(double)*matrixRaw*sourceMatrices.size(), hipMemcpyHostToDevice);

    // GPU kernel function calls
    dim3 dimBlock(32, 32); // Thread block size
    dim3 dimGrid((sourceMatrices.size() + dimBlock.x - 1) 
    / dimBlock.x, (sourceMatrices.size() + dimBlock.y - 1) / dimBlock.y); // Grid Size
    gpuCalc3xMatrixProduct<<<dimGrid, dimBlock>>>
    (dSourceMatrices, dCalcMatrices, dResultMatrices, sourceMatrices.size());

    // Copy results from device memory to host memory
    hipMemcpy(hResultMatrices, dResultMatrices, sizeof(double)*MATRIX3RAW*sourceMatrices.size(), hipMemcpyDeviceToHost);
    
    // Assign the result to a Vector member variable
    resultMatrices.resize(sourceMatrices.size());
    for (int i = 0; i < sourceMatrices.size(); ++i)
    {
        vec.inputVec3d
        (
            hResultMatrices[i*MATRIX3RAW+C1], 
            hResultMatrices[i*MATRIX3RAW+C2], 
            hResultMatrices[i*MATRIX3RAW+C3], 
            i,
            &resultMatrices
        );
    }

    // Release all memory allocated by malloc
    free(hSourceMatrices);
    free(hCalcMatrices);
    free(hResultMatrices);

    hipFree(dSourceMatrices);
    hipFree(dCalcMatrices);
    hipFree(dResultMatrices);
}

void MATRIX::rotTrans(std::vector<VECTOR3D> sourceCoordinates, VECTOR3D rotAngle)
{
    sourceMatrices.resize(sourceCoordinates.size());
    sourceMatrices = sourceCoordinates;

    double calcRotAngle;

    calcRotAngle = rotAngle.x;
    input3xMatrix
    (
        &calcMatrices3x,
        1,    0,                               0, 
        0,    cos(calcRotAngle * PI / 180),    -sin(calcRotAngle * PI / 180),
        0,    sin(calcRotAngle * PI / 180),    cos(calcRotAngle * PI / 180)
    );
    calcMatrix3xProduct();
    sourceMatrices = resultMatrices;

    calcRotAngle = rotAngle.y;
    input3xMatrix
    (
        &calcMatrices3x,
        cos(calcRotAngle * PI / 180),     0,     sin(calcRotAngle * PI / 180), 
        0,                                1,    0,
        -sin(calcRotAngle * PI / 180),    0,     cos(calcRotAngle * PI / 180)
    );
    calcMatrix3xProduct();
    sourceMatrices = resultMatrices;
        
    calcRotAngle = rotAngle.z;
    input3xMatrix
    (
        &calcMatrices3x,
        cos(calcRotAngle * PI / 180),     -sin(calcRotAngle * PI / 180),   0, 
        sin(calcRotAngle * PI / 180),     cos(calcRotAngle * PI / 180),    0,
        0,                                0,                               1
    );
    calcMatrix3xProduct();
    sourceMatrices = resultMatrices;

}

void MATRIX::scaleTrans(std::vector<VECTOR3D> sourceCoordinates, VECTOR3D scalingRate)
{
    sourceMatrices.resize(sourceCoordinates.size());
    sourceMatrices = sourceCoordinates;

    input3xMatrix
    (
        &calcMatrices3x,
        scalingRate.x,  0,              0,
        0,              scalingRate.y,  0,
        0,              0,              scalingRate.z
    );

    calcMatrix3xProduct();
}

__global__ void gpuGetLinePlaneI
(
    double* lineVA, // x1, y1, z1
    double* lineVB, // l, m, n
    double* planeV, // x0, y0, z0
    double* planeN, // p, q, r
    double* lpI,
    int lineAmout,
    int planeAmout
)
{
    // Decide which (i,j) you are in charge of based on your back number
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < lineAmout && j < planeAmout)
    {
        lpI[i*planeAmout*VECTOR3 + j*VECTOR3 + X1] = 
        lineVA[i*VECTOR3 + X1] + 
        ((planeN[j*VECTOR3 + PX] * (-lineVA[i*VECTOR3 + X1] + planeV[j*VECTOR3 + X0])
        + planeN[j*VECTOR3 + QY] * (-lineVA[i*VECTOR3 + Y1] + planeV[j*VECTOR3 + Y0])
        + planeN[j*VECTOR3 + RZ] * (-lineVA[i*VECTOR3 + Z1] + planeV[j*VECTOR3 + Z0]))
        / (lineVB[i*VECTOR3 + X0] * planeN[j*VECTOR3 + PX]
        + lineVB[i*VECTOR3 + Y0] * planeN[j*VECTOR3 + QY]
        + lineVB[i*VECTOR3 + Z0] * planeN[j*VECTOR3 + RZ]))
        * lineVB[i*VECTOR3 + LX];

        lpI[i*planeAmout*VECTOR3 + j*VECTOR3 + Y1] = 
        lineVA[i*VECTOR3 + Y1] + 
        ((planeN[j*VECTOR3 + PX] * (-lineVA[i*VECTOR3 + X1] + planeV[j*VECTOR3 + X0])
        + planeN[j*VECTOR3 + QY] * (-lineVA[i*VECTOR3 + Y1] + planeV[j*VECTOR3 + Y0])
        + planeN[j*VECTOR3 + RZ] * (-lineVA[i*VECTOR3 + Z1] + planeV[j*VECTOR3 + Z0]))
        / (lineVB[i*VECTOR3 + X0] * planeN[j*VECTOR3 + PX]
        + lineVB[i*VECTOR3 + Y0] * planeN[j*VECTOR3 + QY]
        + lineVB[i*VECTOR3 + Z0] * planeN[j*VECTOR3 + RZ]))
        * lineVB[i*VECTOR3 + MY];

        lpI[i*planeAmout*VECTOR3 + j*VECTOR3 + Z1] = 
        lineVA[i*VECTOR3 + Z1] + 
        ((planeN[j*VECTOR3 + PX] * (-lineVA[i*VECTOR3 + X1] + planeV[j*VECTOR3 + X0])
        + planeN[j*VECTOR3 + QY] * (-lineVA[i*VECTOR3 + Y1] + planeV[j*VECTOR3 + Y0])
        + planeN[j*VECTOR3 + RZ] * (-lineVA[i*VECTOR3 + Z1] + planeV[j*VECTOR3 + Z0]))
        / (lineVB[i*VECTOR3 + X0] * planeN[j*VECTOR3 + PX]
        + lineVB[i*VECTOR3 + Y0] * planeN[j*VECTOR3 + QY]
        + lineVB[i*VECTOR3 + Z0] * planeN[j*VECTOR3 + RZ]))
        * lineVB[i*VECTOR3 + NZ];
    } 
}

void EQUATION::getLinePlaneI
(
    std::vector<VECTOR3D> lineVA,
    std::vector<VECTOR3D> lineVB,
    std::vector<VECTOR3D> planeV,
    std::vector<VECTOR3D> planeN
)
{
    int lineAmout = lineVA.size();
    int planeAmout = planeN.size();
    // Allocate memory for each matrix size
    hLineVertexA = (double*)malloc(sizeof(double)*VECTOR3*lineAmout);
    hLineVertexB = (double*)malloc(sizeof(double)*VECTOR3*lineAmout);
    hPlaneVertex = (double*)malloc(sizeof(double)*VECTOR3*planeAmout);
    hPlaneNormal = (double*)malloc(sizeof(double)*VECTOR3*planeAmout);
    hLinePlaneI = (double*)malloc(sizeof(double)*VECTOR3*lineAmout*planeAmout);

    // Copy member variable
    memcpy(hLineVertexA, lineVA.data(), sizeof(double)*VECTOR3*lineAmout);
    memcpy(hLineVertexB, lineVB.data(), sizeof(double)*VECTOR3*lineAmout);
    memcpy(hPlaneVertex, planeV.data(), sizeof(double)*VECTOR3*planeAmout);
    memcpy(hPlaneNormal, planeN.data(), sizeof(double)*VECTOR3*planeAmout);

    // Allocate device-side memory using CUDAMALLOC
    hipMalloc((void**)&dLineVertexA, sizeof(double)*VECTOR3*lineAmout);
    hipMalloc((void**)&dLineVertexB, sizeof(double)*VECTOR3*lineAmout);
    hipMalloc((void**)&dPlaneVertex, sizeof(double)*VECTOR3*planeAmout);
    hipMalloc((void**)&dPlaneNormal, sizeof(double)*VECTOR3*planeAmout);
    hipMalloc((void**)&dLinePlaneI, sizeof(double)*VECTOR3*lineAmout*planeAmout);

    // Copy host-side data to device-side memory
    hipMemcpy(dLineVertexA, hLineVertexA, sizeof(double)*VECTOR3*lineAmout, hipMemcpyHostToDevice);
    hipMemcpy(dLineVertexB, hLineVertexB, sizeof(double)*VECTOR3*lineAmout, hipMemcpyHostToDevice);
    hipMemcpy(dPlaneVertex, hPlaneVertex, sizeof(double)*VECTOR3*planeAmout, hipMemcpyHostToDevice);
    hipMemcpy(dPlaneNormal, hPlaneNormal, sizeof(double)*VECTOR3*planeAmout, hipMemcpyHostToDevice);

    // GPU kernel function calls
    dim3 dimBlock(32, 32); // Thread block size
    dim3 dimGrid((planeAmout + dimBlock.x - 1) 
    / dimBlock.x, (lineAmout + dimBlock.y - 1) / dimBlock.y); // Grid Size
    gpuGetLinePlaneI<<<dimGrid, dimBlock>>>
    (dLineVertexA, dLineVertexB, dPlaneVertex, dPlaneNormal, dLinePlaneI, lineVA.size(), planeN.size());

    // Copy results from device memory to host memory
    hipMemcpy(hLinePlaneI, dLinePlaneI, sizeof(double)*VECTOR3*lineAmout*planeAmout, hipMemcpyDeviceToHost);
    
    // Assign the result to a Vector member variable
    linePlaneI.resize(lineAmout*planeAmout);
    for (int i = 0; i < lineAmout*planeAmout; ++i)
    {
        vec.inputVec3d
        (
            hLinePlaneI[i*VECTOR3+X0], 
            hLinePlaneI[i*VECTOR3+Y0], 
            hLinePlaneI[i*VECTOR3+Z0], 
            i,
            &linePlaneI
        );
    }

    // Release all memory allocated by malloc
    free(hLineVertexA);
    free(hLineVertexB);
    free(hPlaneVertex);
    free(hPlaneNormal);
    free(hLinePlaneI);

    hipFree(dLineVertexA);
    hipFree(dLineVertexB);
    hipFree(dPlaneVertex);
    hipFree(dPlaneNormal);
    hipFree(dLinePlaneI);
}

