#include "hip/hip_runtime.h"
#include "camera.cuh"


void Camera::load(
    std::wstring argName, 
    Vec3d argWPos, 
    Vec3d argRotAngle, 
    double argNearZ, 
    double argFarZ, 
    double argViewAngle, 
    Vec2d argAspectRatio
){
    name = argName;
    wPos = argWPos;
    rotAngle = argRotAngle;
    nearZ = -argNearZ;
    farZ = -argFarZ;
    viewAngle = argViewAngle;
    aspectRatio = argAspectRatio;

    reload = true;
}


void Camera::defineViewVolume(){
    // If no reloading has been done and no changes have been made to the definition, no processing is done.
    if (!reload){
        return;
    }

    // Get screen size
    nearScrSize.x = tan(viewAngle / 2 * PI / 180) * nearZ * 2;
    nearScrSize.y = nearScrSize.x * aspectRatio.y / aspectRatio.x;

    farScrSize.x = nearScrSize.x / 2 * farZ / nearZ;
    farScrSize.y = farScrSize.x * aspectRatio.y / aspectRatio.x;

    viewAngleCos.x = cos(RAD(viewAngle));
    viewAngleCos.y = cos(nearZ / sqrt(nearZ*nearZ + (nearScrSize.y/2) * (nearScrSize.y/2)));

    // Defines the coordinates of the four vertices when the view volume is viewed from the positive y-axis direction.
    viewVolume.xzV[0].x = -nearScrSize.x / 2;
    viewVolume.xzV[1].x  = -farScrSize.x / 2;
    viewVolume.xzV[2].x  = farScrSize.x / 2;
    viewVolume.xzV[3].x  = nearScrSize.x / 2;

    viewVolume.xzV[0].z = -nearZ;
    viewVolume.xzV[1].z = -farZ;
    viewVolume.xzV[2].z = -farZ;
    viewVolume.xzV[3].z = -nearZ;


    // Defines the coordinates of the four vertices when the view volume is viewed from the positive X-axis direction.
    viewVolume.yzV[0].y = nearScrSize.y / 2;
    viewVolume.yzV[1].y = farScrSize.y / 2;
    viewVolume.yzV[2].y = -farScrSize.y / 2;
    viewVolume.yzV[3].y = -nearScrSize.y / 2;

    viewVolume.yzV[0].z = -nearZ;
    viewVolume.yzV[1].z = -farZ;
    viewVolume.yzV[2].z = -farZ;
    viewVolume.yzV[3].z = -nearZ;


    // Defines the coordinates of the vertices in the camera 3D coordinates of the view volume.
    viewVolume.v[RECT_FRONT_TOP_LEFT].x = viewVolume.xzV[0].x;
    viewVolume.v[RECT_FRONT_TOP_LEFT].y = viewVolume.yzV[0].y;
    viewVolume.v[RECT_FRONT_TOP_LEFT].z = -nearZ;

    viewVolume.v[RECT_FRONT_TOP_RIGHT].x = viewVolume.xzV[3].x;
    viewVolume.v[RECT_FRONT_TOP_RIGHT].y = viewVolume.yzV[0].y;
    viewVolume.v[RECT_FRONT_TOP_RIGHT].z = -nearZ;

    viewVolume.v[RECT_FRONT_BOTTOM_RIGHT].x = viewVolume.xzV[3].x;
    viewVolume.v[RECT_FRONT_BOTTOM_RIGHT].y = viewVolume.yzV[3].y;
    viewVolume.v[RECT_FRONT_BOTTOM_RIGHT].z = -nearZ;

    viewVolume.v[RECT_FRONT_BOTTOM_LEFT].x = viewVolume.xzV[0].x;
    viewVolume.v[RECT_FRONT_BOTTOM_LEFT].y = viewVolume.yzV[3].y;
    viewVolume.v[RECT_FRONT_BOTTOM_LEFT].z = -nearZ;

    viewVolume.v[RECT_BACK_TOP_LEFT].x = viewVolume.xzV[1].x;
    viewVolume.v[RECT_BACK_TOP_LEFT].y = viewVolume.yzV[1].y;
    viewVolume.v[RECT_BACK_TOP_LEFT].z = -farZ;

    viewVolume.v[RECT_BACK_TOP_RIGHT].x = viewVolume.xzV[2].x;
    viewVolume.v[RECT_BACK_TOP_RIGHT].y = viewVolume.yzV[1].y;
    viewVolume.v[RECT_BACK_TOP_RIGHT].z = -farZ;

    viewVolume.v[RECT_BACK_BOTTOM_RIGHT].x = viewVolume.xzV[2].x;
    viewVolume.v[RECT_BACK_BOTTOM_RIGHT].y = viewVolume.yzV[2].y;
    viewVolume.v[RECT_BACK_BOTTOM_RIGHT].z = -farZ;

    viewVolume.v[RECT_BACK_BOTTOM_LEFT].x = viewVolume.xzV[1].x;
    viewVolume.v[RECT_BACK_BOTTOM_LEFT].y = viewVolume.yzV[2].y;
    viewVolume.v[RECT_BACK_BOTTOM_LEFT].z = -farZ;


    // Stores the coordinates of the vertices that are the start and end points of each line segment of the view volume.
    viewVolume.lines[0].startV = viewVolume.v[RECT_L1_STARTV];
    viewVolume.lines[1].startV = viewVolume.v[RECT_L2_STARTV];
    viewVolume.lines[2].startV = viewVolume.v[RECT_L3_STARTV];
    viewVolume.lines[3].startV = viewVolume.v[RECT_L4_STARTV];
    viewVolume.lines[4].startV = viewVolume.v[RECT_L5_STARTV];
    viewVolume.lines[5].startV = viewVolume.v[RECT_L6_STARTV];
    viewVolume.lines[6].startV = viewVolume.v[RECT_L7_STARTV];
    viewVolume.lines[7].startV = viewVolume.v[RECT_L8_STARTV];
    viewVolume.lines[8].startV = viewVolume.v[RECT_L9_STARTV];
    viewVolume.lines[9].startV = viewVolume.v[RECT_L10_STARTV];
    viewVolume.lines[10].startV = viewVolume.v[RECT_L11_STARTV];
    viewVolume.lines[11].startV = viewVolume.v[RECT_L12_STARTV];

    viewVolume.lines[0].endV = viewVolume.v[RECT_L1_ENDV];
    viewVolume.lines[1].endV = viewVolume.v[RECT_L2_ENDV];
    viewVolume.lines[2].endV = viewVolume.v[RECT_L3_ENDV];
    viewVolume.lines[3].endV = viewVolume.v[RECT_L4_ENDV];
    viewVolume.lines[4].endV = viewVolume.v[RECT_L5_ENDV];
    viewVolume.lines[5].endV = viewVolume.v[RECT_L6_ENDV];
    viewVolume.lines[6].endV = viewVolume.v[RECT_L7_ENDV];
    viewVolume.lines[7].endV = viewVolume.v[RECT_L8_ENDV];
    viewVolume.lines[8].endV = viewVolume.v[RECT_L9_ENDV];
    viewVolume.lines[9].endV = viewVolume.v[RECT_L10_ENDV];
    viewVolume.lines[10].endV = viewVolume.v[RECT_L11_ENDV];
    viewVolume.lines[11].endV = viewVolume.v[RECT_L12_ENDV];


    // Obtain a 3D vector for each line segment.
    for (auto& i : viewVolume.lines){
        i.vec.x = i.endV.x - i.startV.x;
        i.vec.y = i.endV.y - i.startV.y;
        i.vec.z = i.endV.z - i.startV.z;
    }

    // Stores the coordinates of the starting vertex of the normal vector for each face of the view volume.
    viewVolume.face.v[SURFACE_TOP] = viewVolume.v[RECT_FRONT_TOP_LEFT];
    viewVolume.face.v[SURFACE_FRONT] = viewVolume.v[RECT_FRONT_TOP_LEFT];
    viewVolume.face.v[SURFACE_RIGHT] = viewVolume.v[RECT_BACK_BOTTOM_RIGHT];
    viewVolume.face.v[SURFACE_LEFT] = viewVolume.v[RECT_FRONT_TOP_LEFT];
    viewVolume.face.v[SURFACE_BACK] = viewVolume.v[RECT_BACK_BOTTOM_RIGHT];
    viewVolume.face.v[SURFACE_BOTTOM] = viewVolume.v[RECT_BACK_BOTTOM_RIGHT];


    // Get the normal vector of each face of the view volume.
    std::vector<Vec3d> calcVA(6);
    std::vector<Vec3d> calcVB(6);

    calcVA[SURFACE_TOP] = viewVolume.lines[0].vec;
    calcVB[SURFACE_TOP] = viewVolume.lines[4].vec;

    calcVA[SURFACE_FRONT] = viewVolume.lines[0].vec;
    calcVB[SURFACE_FRONT] = viewVolume.lines[1].vec;

    calcVA[SURFACE_RIGHT] = viewVolume.lines[1].vec;
    calcVB[SURFACE_RIGHT] = viewVolume.lines[5].vec;

    calcVA[SURFACE_LEFT] = viewVolume.lines[3].vec;
    calcVB[SURFACE_LEFT] = viewVolume.lines[4].vec;

    calcVA[SURFACE_BACK] = viewVolume.lines[8].vec;
    calcVB[SURFACE_BACK] = viewVolume.lines[9].vec;

    calcVA[SURFACE_BOTTOM] = viewVolume.lines[2].vec;
    calcVB[SURFACE_BOTTOM] = viewVolume.lines[6].vec;

    for (int i = 0; i < 6; i++){
        viewVolume.face.normal[i].x = calcVA[i].y * calcVB[i].z - calcVA[i].z * calcVB[i].y;
        viewVolume.face.normal[i].y = calcVA[i].z * calcVB[i].x - calcVA[i].x * calcVB[i].z;
        viewVolume.face.normal[i].z = calcVA[i].x * calcVB[i].y - calcVA[i].y * calcVB[i].x;
    }

    reload = false;

}


void Camera::objCulling(std::unordered_map<std::wstring, Object> objects){
    std::vector<Vec3d> rangeVs(objects.size() * 8);

    int iN1 = 0;
    for (auto obj : objects){
        for (int i = 0; i < 8; i++){
            rangeVs[iN1*8 + i] = obj.second.range.wVertex[i];
        }
        iN1 += 1;
    }

    rangeVs = mt.transRotConvert(wPos, rotAngle, rangeVs);

    std::vector<std::wstring> objOrder;
    std::vector<Vec3d> oppositeSideXzVs;
    std::vector<Vec3d> oppositeSideYzVs;
    std::vector<Vec3d> oppositeSideVs;
    std::vector<double> orizinZ;
    std::vector<double> oppositeZ;

    std::vector<Vec3d> rectVs(2);
    bool status = false;
    Vec3d pushVec1;

    int iN2 = 0;

    for (auto obj : objects){
        objOrder.push_back(obj.first);
        
        status = false;
        for (int i = 0; i < 8; i++){
            if (status){
                if (rangeVs[iN2*8 + i].x < rectVs[0].x){
                    rectVs[0].x = rangeVs[iN2*8 + i].x;
                }
                if (rangeVs[iN2*8 + i].y < rectVs[0].y){
                    rectVs[0].y = rangeVs[iN2*8 + i].y;
                }
                if (rangeVs[iN2*8 + i].z > rectVs[0].z){
                    rectVs[0].z = rangeVs[iN2*8 + i].z;
                }

                // Processing with respect to opposite point
                if (rangeVs[iN2*8 + i].x > rectVs[1].x){
                    rectVs[1].x = rangeVs[iN2*8 + i].x;
                }
                if (rangeVs[iN2*8 + i].y > rectVs[1].y){
                    rectVs[1].y = rangeVs[iN2*8 + i].y;
                }
                if (rangeVs[iN2*8 + i].z < rectVs[1].z){
                    rectVs[1].z = rangeVs[iN2*8 + i].z;
                }
            }
            else{
                rectVs[0].x = rangeVs[iN2*8 + i].x;
                rectVs[0].y = rangeVs[iN2*8 + i].y;
                rectVs[0].z = rangeVs[iN2*8 + i].z;

                rectVs[1].x = rangeVs[iN2*8 + i].x;
                rectVs[1].y = rangeVs[iN2*8 + i].y;
                rectVs[1].z = rangeVs[iN2*8 + i].z;
                status = true;
            }
        }
        iN2 += 1;
        
        pushVec1 = {rectVs[0].x, 0, rectVs[1].z};
        oppositeSideVs.push_back(pushVec1);

        pushVec1 = {rectVs[1].x, 0, rectVs[1].z};
        oppositeSideVs.push_back(pushVec1);

        pushVec1 = {0, rectVs[0].y, rectVs[1].z};
        oppositeSideVs.push_back(pushVec1);

        pushVec1 = {0, rectVs[1].y, rectVs[1].z};
        oppositeSideVs.push_back(pushVec1);

        orizinZ.push_back(rectVs[0].z);
        oppositeZ.push_back(rectVs[1].z);
    }

    Vec3d zVec = {0, 0, -1};

    std::vector<double> rangeXyzVsCos = vec.getVecsDotCos(zVec, oppositeSideVs);

    for (int i = 0; i < rangeXyzVsCos.size() / 4; i++){
        if (orizinZ[i] >= farZ && oppositeZ[i] <= nearZ){
            if (rangeXyzVsCos[i*4] >= viewAngleCos.x || rangeXyzVsCos[i*4 + 1] >= viewAngleCos.x){
                if (rangeXyzVsCos[i*4 + 2] >= viewAngleCos.y || rangeXyzVsCos[i*4 + 3] >= viewAngleCos.y){
                    renderTargetObj.push_back(objOrder[i]);
                }
            }
        }
    }
}


void Camera::polyBilateralJudge(std::unordered_map<std::wstring, Object> objects){
    std::vector<Vec3d> vs;
    std::vector<Vec3d> normals;

    std::vector<int> objFaceIs;

    FaceNormals faceN;
    int iN1 = 0;
    for(int i = 0; i < renderTargetObj.size(); i++){
        for (int j = 0; j < objects[renderTargetObj[i]].poly.vId.size(); j++){
            faceN.v.push_back(
                objects[renderTargetObj[i]].v.world[
                    objects[renderTargetObj[i]].poly.vId[j].n1
                ]
            );

            faceN.normal.push_back(
                objects[renderTargetObj[i]].v.normal[
                    objects[renderTargetObj[i]].poly.normalId[j].n1
                ]
            );

            iN1 += 1;
        }

        objFaceIs.push_back(iN1);
        iN1 = 0;

        vs.insert(vs.end(), faceN.v.begin(), faceN.v.end());
        normals.insert(normals.end(), faceN.normal.begin(), faceN.normal.end());
        faceN.v.clear();
        faceN.normal.clear();
    }

    std::vector<Vec3d> cnvtVs = mt.transRotConvert(wPos, rotAngle, vs);
    std::vector<Vec3d> cnvtNs = mt.rotConvert(rotAngle, normals);

    std::vector<double> vecsCos = vec.getSameSizeVecsDotCos(cnvtVs, cnvtNs);

    int iN2 = 0;
    PolyNameInfo pushPoly;
    for (int i = 0; i < objFaceIs.size(); i++){
        for (int j = 0; j < objFaceIs[i]; j++){
            if (vecsCos[iN2 + j] <= 0){
                pushPoly.objName = renderTargetObj[i];
                pushPoly.polyId = j;

                renderTargetPoly.push_back(pushPoly);
            }
        }

        iN2 += objFaceIs[i];
    }
}


void Camera::polyCulling(
    std::unordered_map<std::wstring, Object> objects, std::vector<RasterizeSource>* ptRS
){
    std::vector<Vec3d> polyVs;
    std::vector<Vec3d> polyNs;
    for (int i = 0; i < renderTargetPoly.size(); i++){
        polyVs.push_back(
            objects[renderTargetPoly[i].objName].v.world[
                objects[renderTargetPoly[i].objName].poly.vId[renderTargetPoly[i].polyId].n1
            ]
        );
        polyVs.push_back(
            objects[renderTargetPoly[i].objName].v.world[
                objects[renderTargetPoly[i].objName].poly.vId[renderTargetPoly[i].polyId].n2
            ]
        );
        polyVs.push_back(
            objects[renderTargetPoly[i].objName].v.world[
                objects[renderTargetPoly[i].objName].poly.vId[renderTargetPoly[i].polyId].n3
            ]
        );

        polyNs.push_back(
            objects[renderTargetPoly[i].objName].v.normal[
                objects[renderTargetPoly[i].objName].poly.normalId[renderTargetPoly[i].polyId].n1
            ]
        );

    }

    std::vector<Vec3d> cnvtPolyVs = mt.transRotConvert(wPos, rotAngle, polyVs);
    std::vector<Vec3d> cnvtPolyNs = mt.rotConvert(rotAngle, polyNs);

    std::vector<Vec3d> cnvt2dPolyVs;
    for (int i = 0; i < cnvtPolyVs.size(); i++){
        cnvt2dPolyVs.push_back({
            cnvtPolyVs[i].x,
            0,
            cnvtPolyVs[i].z
        });

        cnvt2dPolyVs.push_back({
            0,
            cnvtPolyVs[i].y,
            cnvtPolyVs[i].z
        });
    }

    Vec3d zVec = {0, 0, -1};

    std::vector<double> polyVCos = vec.getVecsDotCos(zVec, cnvt2dPolyVs);

    RasterizeSource pushRS;
    std::vector<PolyNameInfo> needRangeVs;
    std::vector<int> cnvtVsIndex;
    int inViewVolume = 0;
    for (int i = 0; i < renderTargetPoly.size(); i++){
        if (cnvtPolyVs[i*3].z >= farZ && cnvtPolyVs[i*3].z <= nearZ){
            if (polyVCos[i*6] >= viewAngleCos.x){
                if (polyVCos[i*6 + 1] >= viewAngleCos.y){
                    inViewVolume += 1;
                    pushRS.scPixelVs.wVs.push_back(cnvtPolyVs[i*3]);
                }
            }
        }

        if (cnvtPolyVs[i*3 + 1].z >= farZ && cnvtPolyVs[i*3 + 1].z <= nearZ){
            if (polyVCos[i*6 + 2] >= viewAngleCos.x){
                if (polyVCos[i*6 + 3] >= viewAngleCos.y){
                    inViewVolume += 1;
                    pushRS.scPixelVs.wVs.push_back(cnvtPolyVs[i*3 + 1]);
                }
            }
        }

        if (cnvtPolyVs[i*3 + 2].z >= farZ && cnvtPolyVs[i*3 + 2].z <= nearZ){
            if (polyVCos[i*6 + 4] >= viewAngleCos.x){
                if (polyVCos[i*6 + 5] >= viewAngleCos.y){
                    inViewVolume += 1;
                    pushRS.scPixelVs.wVs.push_back(cnvtPolyVs[i*3 + 2]);
                }
            }
        }

        if (inViewVolume != 0){
            pushRS.renderPoly.objName = renderTargetPoly[i].objName;
            pushRS.renderPoly.polyId = renderTargetPoly[i].polyId;

            pushRS.polyCamVs.push_back(cnvtPolyVs[i*3]);
            pushRS.polyCamVs.push_back(cnvtPolyVs[i*3 + 1]);
            pushRS.polyCamVs.push_back(cnvtPolyVs[i*3 + 2]);

            pushRS.polyN = cnvtPolyNs[i];

            (*ptRS).push_back(pushRS);

            if (inViewVolume != 3){
                shapeCnvtTargetI.push_back((*ptRS).size() - 1);
            }

            inViewVolume = 0;
            pushRS.polyCamVs.clear();
            pushRS.scPixelVs.wVs.clear();
        }
        else{
            needRangeVs.push_back({
                renderTargetPoly[i].objName, renderTargetPoly[i].polyId
            });

            cnvtVsIndex.push_back(i);
        }
    }

    RangeRect polyRange;
    std::vector<Vec3d> oppositeSideVs;
    std::vector<double> orizinZ;
    std::vector<double> oppositeZ;
    Vec3d pushVec;
    for (int i = 0; i < needRangeVs.size(); i++){
        polyRange.origin.x = GLPA_CAMERA_OBJ_WV_1(n1).x;
        polyRange.origin.y = GLPA_CAMERA_OBJ_WV_1(n1).y;
        polyRange.origin.z = GLPA_CAMERA_OBJ_WV_1(n1).z;

        polyRange.opposite.x = GLPA_CAMERA_OBJ_WV_1(n1).x;
        polyRange.opposite.y = GLPA_CAMERA_OBJ_WV_1(n1).y;
        polyRange.opposite.z = GLPA_CAMERA_OBJ_WV_1(n1).z;
        polyRange.status = true;

        GLPA_CAMERA_POLY_NEED_RANGE_IFS(n2);
        GLPA_CAMERA_POLY_NEED_RANGE_IFS(n3);

        pushVec = {polyRange.origin.x, 0, polyRange.opposite.z};
        oppositeSideVs.push_back(pushVec);

        pushVec = {polyRange.opposite.x, 0, polyRange.opposite.z};
        oppositeSideVs.push_back(pushVec);

        pushVec = {0, polyRange.origin.y, polyRange.opposite.z};
        oppositeSideVs.push_back(pushVec);

        pushVec = {0, polyRange.origin.y, polyRange.opposite.z};
        oppositeSideVs.push_back(pushVec);

        orizinZ.push_back(polyRange.origin.z);
        oppositeZ.push_back(polyRange.opposite.z);
    }

    std::vector<double> rangeXyzVsCos = vec.getVecsDotCos(zVec, oppositeSideVs);

    RasterizeSource pushRS2;
    for (int i = 0; i < needRangeVs.size(); i++){
        if (orizinZ[i] >= farZ && oppositeZ[i] <= nearZ){
            if (rangeXyzVsCos[i*4] >= viewAngleCos.x || rangeXyzVsCos[i*4 + 1] >= viewAngleCos.x){
                if (rangeXyzVsCos[i*4 + 2] >= viewAngleCos.y || rangeXyzVsCos[i*4 + 3] >= viewAngleCos.y){
                    pushRS2.renderPoly.objName = needRangeVs[i].objName;
                    pushRS2.renderPoly.polyId = needRangeVs[i].polyId;

                    pushRS2.polyCamVs.push_back(cnvtPolyVs[cnvtVsIndex[i]*3]);
                    pushRS2.polyCamVs.push_back(cnvtPolyVs[cnvtVsIndex[i]*3 + 1]);
                    pushRS2.polyCamVs.push_back(cnvtPolyVs[cnvtVsIndex[i]*3 + 2]);

                    pushRS2.polyN = cnvtPolyNs[cnvtVsIndex[i]];

                    (*ptRS).push_back(pushRS2);
                    shapeCnvtTargetI.push_back((*ptRS).size() - 1);
                    
                    pushRS2.polyCamVs.clear();
                }
            }
        }
    }
}

void Camera::polyShapeConvert(
    std::unordered_map<std::wstring, Object> objects, std::vector<RasterizeSource> *ptRS
){
    std::vector<Vec3d> polyOneVs;
    std::vector<Vec3d> polyNs;
    std::vector<Vec3d> vvLineStartVs;
    std::vector<Vec3d> vvLineEndVs;

    for (int i = 0; i < shapeCnvtTargetI.size(); i++){
        polyOneVs.push_back((*ptRS)[shapeCnvtTargetI[i]].polyCamVs[0]);
        polyNs.push_back((*ptRS)[shapeCnvtTargetI[i]].polyN);
    }

    for (int i = 0; i < viewVolume.lines.size(); i++){
        vvLineStartVs.push_back(viewVolume.lines[i].startV);
        vvLineEndVs.push_back(viewVolume.lines[i].endV);
    }

    std::vector<Vec3d> polyLineStartVs;
    std::vector<Vec3d> polyLineEndVs;

    for (int i = 0; i < shapeCnvtTargetI.size(); i++){
        for (int j = 0; j < 3; j++){
            polyLineStartVs.push_back(
                (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[j]
            );
        }

        for (int j = 0; j < 3; j++){
            if (j != 2){
                polyLineEndVs.push_back(
                    (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[j + 1]
                );
            }
            else{
                polyLineEndVs.push_back(
                    (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[0]
                );
            }
        }
    }

    int polyFaceAmount = shapeCnvtTargetI.size();
    int vvLineAmout = 12;

    int vvFaceAmout = 6;
    int polyLineAmout = shapeCnvtTargetI.size() * 3;

    double* hPolyFaceDot = (double*)malloc(sizeof(double)*polyFaceAmount*vvLineAmout*2);
    double* hVvFaceDot = (double*)malloc(sizeof(double)*vvFaceAmout*polyLineAmout*2);

    double* hPolyOneVs = (double*)malloc(sizeof(double)*shapeCnvtTargetI.size()*3);
    double* hPolyNs = (double*)malloc(sizeof(double)*shapeCnvtTargetI.size()*3);
    double* hVvLineStartVs = (double*)malloc(sizeof(double)*viewVolume.lines.size()*3);
    double* hVvLineEndVs = (double*)malloc(sizeof(double)*viewVolume.lines.size()*3);

    for (int i = 0; i < shapeCnvtTargetI.size(); i++){
        hPolyOneVs[i*3] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[0].x;
        hPolyOneVs[i*3 + 1] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[0].y;
        hPolyOneVs[i*3 + 2] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[0].z;

        hPolyNs[i*3] = (*ptRS)[shapeCnvtTargetI[i]].polyN.x;
        hPolyNs[i*3 + 1] = (*ptRS)[shapeCnvtTargetI[i]].polyN.x;
        hPolyNs[i*3 + 2] = (*ptRS)[shapeCnvtTargetI[i]].polyN.x;
    }
    
    for (int i = 0; i < viewVolume.lines.size(); i++){
        hVvLineStartVs[i*3] = viewVolume.lines[i].startV.x;
        hVvLineStartVs[i*3 + 1] = viewVolume.lines[i].startV.y;
        hVvLineStartVs[i*3 + 2] = viewVolume.lines[i].startV.z;

        hVvLineEndVs[i*3] = viewVolume.lines[i].endV.x;
        hVvLineEndVs[i*3 + 1] = viewVolume.lines[i].endV.y;
        hVvLineEndVs[i*3 + 2] = viewVolume.lines[i].endV.z;
    }


    double* hVvOneVs = (double*)malloc(sizeof(double)*viewVolume.face.v.size()*3);
    double* hVvNs = (double*)malloc(sizeof(double)*viewVolume.face.normal.size()*3);
    double* hPolyLineStartVs = (double*)malloc(sizeof(double)*shapeCnvtTargetI.size()*3*3);
    double* hPolyLineEndVs = (double*)malloc(sizeof(double)*shapeCnvtTargetI.size()*3*3);
    for (int i = 0; i < shapeCnvtTargetI.size(); i++){
        for (int j = 0; j < 3; j++){
            hPolyLineStartVs[i*3 + j*3] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[j].x;
            hPolyLineStartVs[i*3 + j*3 + 1] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[j].y;
            hPolyLineStartVs[i*3 + j*3 + 2] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[j].z;
        }

        for (int j = 0; j < 3; j++){
            if (j != 2){
                hPolyLineEndVs[i*3 + j*3] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[j + 1].x;
                hPolyLineEndVs[i*3 + j*3 + 1] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[j + 1].y;
                hPolyLineEndVs[i*3 + j*3 + 2] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[j + 1].z;
            }
            else{
                polyLineEndVs.push_back(
                    (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[0]
                );
                hPolyLineEndVs[i*3 + j*3] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[0].x;
                hPolyLineEndVs[i*3 + j*3 + 1] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[0].y;
                hPolyLineEndVs[i*3 + j*3 + 2] = (*ptRS)[shapeCnvtTargetI[i]].polyCamVs[0].z;
            }
        }
    }


    double* dPolyFaceDot;
    double* dVvFaceDot;
    hipMalloc((void**)&dPolyFaceDot, sizeof(double)*polyFaceAmount*vvLineAmout*2);
    hipMalloc((void**)&dVvFaceDot, sizeof(double)*vvFaceAmout*polyLineAmout*2);

    double* dPolyOneVs;
    double* dPolyNs;
    double* dVvLineStartVs;
    double* dVvLineEndVs;
    hipMalloc((void**)&dPolyOneVs, sizeof(double)*polyOneVs.size()*3);
    hipMalloc((void**)&dPolyNs, sizeof(double)*polyNs.size()*3);
    hipMalloc((void**)&dVvLineStartVs, sizeof(double)*vvLineStartVs.size()*3);
    hipMalloc((void**)&dVvLineEndVs, sizeof(double)*vvLineEndVs.size()*3);

    hipMemcpy(dPolyOneVs, hPolyOneVs, sizeof(double)*polyOneVs.size()*3, hipMemcpyHostToDevice);
    hipMemcpy(dPolyNs, hPolyNs, sizeof(double)*polyNs.size()*3, hipMemcpyHostToDevice);
    hipMemcpy(dVvLineStartVs, hVvLineStartVs, sizeof(double)*vvLineStartVs.size()*3, hipMemcpyHostToDevice);
    hipMemcpy(dVvLineEndVs, hVvLineEndVs, sizeof(double)*vvLineEndVs.size()*3, hipMemcpyHostToDevice);

    double* dVvOneVs;
    double* dVvNs;
    double* dPolyLineStartVs;
    double* dPolyLineEndVs;
    hipMalloc((void**)&dVvOneVs, sizeof(double)*viewVolume.face.v.size()*3);
    hipMalloc((void**)&dVvNs, sizeof(double)*viewVolume.face.normal.size()*3);
    hipMalloc((void**)&dPolyLineStartVs, sizeof(double)*polyLineStartVs.size()*3);
    hipMalloc((void**)&dPolyLineEndVs, sizeof(double)*polyLineEndVs.size()*3);

    hipMemcpy(dVvOneVs, hVvOneVs, sizeof(double)*viewVolume.face.v.size()*3, hipMemcpyHostToDevice);
    hipMemcpy(dVvNs, hVvNs, sizeof(double)*viewVolume.face.normal.size()*3, hipMemcpyHostToDevice);
    hipMemcpy(dPolyLineStartVs, hPolyLineStartVs, sizeof(double)*polyLineStartVs.size()*3, hipMemcpyHostToDevice);
    hipMemcpy(dPolyLineEndVs, hPolyLineEndVs, sizeof(double)*polyLineEndVs.size()*3, hipMemcpyHostToDevice);

    dim3 dimBlock(32, 32); // Thread block size
    dim3 dimGrid((polyFaceAmount + dimBlock.x - 1) 
    / dimBlock.x, (polyFaceAmount + dimBlock.y - 1) / dimBlock.y); // Grid Size
    glpaGpuGetPolyVvDot<<<dimGrid, dimBlock>>>(
        dPolyFaceDot,
        dVvFaceDot,
        dPolyOneVs,
        dPolyNs,
        dVvLineStartVs,
        dVvLineEndVs,
        dVvOneVs,
        dVvNs,
        dPolyLineStartVs,
        dPolyLineEndVs,
        polyFaceAmount
    );

    hipMemcpy(hPolyFaceDot, dPolyFaceDot, sizeof(double)*polyFaceAmount*vvLineAmout*2, hipMemcpyDeviceToHost);
    hipMemcpy(hVvFaceDot, dVvFaceDot, sizeof(double)*vvFaceAmout*polyLineAmout*2, hipMemcpyDeviceToHost);



    

    


}
