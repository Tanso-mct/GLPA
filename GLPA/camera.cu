#include "hip/hip_runtime.h"
#include "camera.cuh"


void Camera::load(
    std::wstring argName, 
    Vec3d argWPos, 
    Vec3d argRotAngle, 
    double argNearZ, 
    double argFarZ, 
    double argViewAngle, 
    Vec2d argAspectRatio
){
    name = argName;
    wPos = argWPos;
    rotAngle = argRotAngle;
    nearZ = argNearZ;
    farZ = argFarZ;
    viewAngle = argViewAngle;
    aspectRatio = argAspectRatio;

    reload = true;
}


void Camera::defineViewVolume(){
    // If no reloading has been done and no changes have been made to the definition, no processing is done.
    if (!reload){
        return;
    }

    // Get screen size
    nearScrSize.x = tan(viewAngle / 2 * PI / 180) * nearZ * 2;
    nearScrSize.y = nearScrSize.x * aspectRatio.y / aspectRatio.x;

    farScrSize.x = nearScrSize.x / 2 * farZ / nearZ;
    farScrSize.y = farScrSize.x * aspectRatio.y / aspectRatio.x;


    // Defines the coordinates of the four vertices when the view volume is viewed from the positive y-axis direction.
    viewVolume.xzV[0].x = -nearScrSize.x / 2;
    viewVolume.xzV[1].x  = -farScrSize.x / 2;
    viewVolume.xzV[2].x  = farScrSize.x / 2;
    viewVolume.xzV[3].x  = nearScrSize.x / 2;

    viewVolume.xzV[0].z = -nearZ;
    viewVolume.xzV[1].z = -farZ;
    viewVolume.xzV[2].z = -farZ;
    viewVolume.xzV[3].z = -nearZ;


    // Defines the coordinates of the four vertices when the view volume is viewed from the positive X-axis direction.
    viewVolume.yzV[0].y = nearScrSize.y / 2;
    viewVolume.yzV[1].y = farScrSize.y / 2;
    viewVolume.yzV[2].y = -farScrSize.y / 2;
    viewVolume.yzV[3].y = -nearScrSize.y / 2;

    viewVolume.yzV[0].z = -nearZ;
    viewVolume.yzV[1].z = -farZ;
    viewVolume.yzV[2].z = -farZ;
    viewVolume.yzV[3].z = -nearZ;


    // Defines the coordinates of the vertices in the camera 3D coordinates of the view volume.
    viewVolume.v[RECT_FRONT_TOP_LEFT].x = viewVolume.xzV[0].x;
    viewVolume.v[RECT_FRONT_TOP_LEFT].y = viewVolume.yzV[0].y;
    viewVolume.v[RECT_FRONT_TOP_LEFT].z = -nearZ;

    viewVolume.v[RECT_FRONT_TOP_RIGHT].x = viewVolume.xzV[3].x;
    viewVolume.v[RECT_FRONT_TOP_RIGHT].y = viewVolume.yzV[0].y;
    viewVolume.v[RECT_FRONT_TOP_RIGHT].z = -nearZ;

    viewVolume.v[RECT_FRONT_BOTTOM_RIGHT].x = viewVolume.xzV[3].x;
    viewVolume.v[RECT_FRONT_BOTTOM_RIGHT].y = viewVolume.yzV[3].y;
    viewVolume.v[RECT_FRONT_BOTTOM_RIGHT].z = -nearZ;

    viewVolume.v[RECT_FRONT_BOTTOM_LEFT].x = viewVolume.xzV[0].x;
    viewVolume.v[RECT_FRONT_BOTTOM_LEFT].y = viewVolume.yzV[3].y;
    viewVolume.v[RECT_FRONT_BOTTOM_LEFT].z = -nearZ;

    viewVolume.v[RECT_BACK_TOP_LEFT].x = viewVolume.xzV[1].x;
    viewVolume.v[RECT_BACK_TOP_LEFT].y = viewVolume.yzV[1].y;
    viewVolume.v[RECT_BACK_TOP_LEFT].z = -farZ;

    viewVolume.v[RECT_BACK_TOP_RIGHT].x = viewVolume.xzV[2].x;
    viewVolume.v[RECT_BACK_TOP_RIGHT].y = viewVolume.yzV[1].y;
    viewVolume.v[RECT_BACK_TOP_RIGHT].z = -farZ;

    viewVolume.v[RECT_BACK_BOTTOM_RIGHT].x = viewVolume.xzV[2].x;
    viewVolume.v[RECT_BACK_BOTTOM_RIGHT].y = viewVolume.yzV[2].y;
    viewVolume.v[RECT_BACK_BOTTOM_RIGHT].z = -farZ;

    viewVolume.v[RECT_BACK_BOTTOM_LEFT].x = viewVolume.xzV[1].x;
    viewVolume.v[RECT_BACK_BOTTOM_LEFT].y = viewVolume.yzV[2].y;
    viewVolume.v[RECT_BACK_BOTTOM_LEFT].z = -farZ;


    // Stores the coordinates of the vertices that are the start and end points of each line segment of the view volume.
    viewVolume.lines[0].startV = viewVolume.v[RECT_L1_STARTV];
    viewVolume.lines[1].startV = viewVolume.v[RECT_L2_STARTV];
    viewVolume.lines[2].startV = viewVolume.v[RECT_L3_STARTV];
    viewVolume.lines[3].startV = viewVolume.v[RECT_L4_STARTV];
    viewVolume.lines[4].startV = viewVolume.v[RECT_L5_STARTV];
    viewVolume.lines[5].startV = viewVolume.v[RECT_L6_STARTV];
    viewVolume.lines[6].startV = viewVolume.v[RECT_L7_STARTV];
    viewVolume.lines[7].startV = viewVolume.v[RECT_L8_STARTV];
    viewVolume.lines[8].startV = viewVolume.v[RECT_L9_STARTV];
    viewVolume.lines[9].startV = viewVolume.v[RECT_L10_STARTV];
    viewVolume.lines[10].startV = viewVolume.v[RECT_L11_STARTV];
    viewVolume.lines[11].startV = viewVolume.v[RECT_L12_STARTV];

    viewVolume.lines[0].endV = viewVolume.v[RECT_L1_ENDV];
    viewVolume.lines[1].endV = viewVolume.v[RECT_L2_ENDV];
    viewVolume.lines[2].endV = viewVolume.v[RECT_L3_ENDV];
    viewVolume.lines[3].endV = viewVolume.v[RECT_L4_ENDV];
    viewVolume.lines[4].endV = viewVolume.v[RECT_L5_ENDV];
    viewVolume.lines[5].endV = viewVolume.v[RECT_L6_ENDV];
    viewVolume.lines[6].endV = viewVolume.v[RECT_L7_ENDV];
    viewVolume.lines[7].endV = viewVolume.v[RECT_L8_ENDV];
    viewVolume.lines[8].endV = viewVolume.v[RECT_L9_ENDV];
    viewVolume.lines[9].endV = viewVolume.v[RECT_L10_ENDV];
    viewVolume.lines[10].endV = viewVolume.v[RECT_L11_ENDV];
    viewVolume.lines[11].endV = viewVolume.v[RECT_L12_ENDV];


    // Obtain a 3D vector for each line segment.
    for (auto& i : viewVolume.lines){
        i.vec.x = i.endV.x - i.startV.x;
        i.vec.y = i.endV.y - i.startV.y;
        i.vec.z = i.endV.z - i.startV.z;
    }

    // Stores the coordinates of the starting vertex of the normal vector for each face of the view volume.
    viewVolume.face[SURFACE_TOP].v = viewVolume.v[RECT_FRONT_TOP_LEFT];
    viewVolume.face[SURFACE_FRONT].v = viewVolume.v[RECT_FRONT_TOP_LEFT];
    viewVolume.face[SURFACE_RIGHT].v = viewVolume.v[RECT_BACK_BOTTOM_RIGHT];
    viewVolume.face[SURFACE_LEFT].v = viewVolume.v[RECT_FRONT_TOP_LEFT];
    viewVolume.face[SURFACE_BACK].v = viewVolume.v[RECT_BACK_BOTTOM_RIGHT];
    viewVolume.face[SURFACE_BOTTOM].v = viewVolume.v[RECT_BACK_BOTTOM_RIGHT];


    // Get the normal vector of each face of the view volume.
    std::vector<Vec3d> calcVA(6);
    std::vector<Vec3d> calcVB(6);

    calcVA[SURFACE_TOP] = viewVolume.lines[0].vec;
    calcVB[SURFACE_TOP] = viewVolume.lines[4].vec;

    calcVA[SURFACE_FRONT] = viewVolume.lines[0].vec;
    calcVB[SURFACE_FRONT] = viewVolume.lines[1].vec;

    calcVA[SURFACE_RIGHT] = viewVolume.lines[1].vec;
    calcVB[SURFACE_RIGHT] = viewVolume.lines[5].vec;

    calcVA[SURFACE_LEFT] = viewVolume.lines[3].vec;
    calcVB[SURFACE_LEFT] = viewVolume.lines[4].vec;

    calcVA[SURFACE_BACK] = viewVolume.lines[8].vec;
    calcVB[SURFACE_BACK] = viewVolume.lines[9].vec;

    calcVA[SURFACE_BOTTOM] = viewVolume.lines[2].vec;
    calcVB[SURFACE_BOTTOM] = viewVolume.lines[6].vec;

    for (int i = 0; i < 6; i++){
        viewVolume.face[i].normal.x = calcVA[i].y * calcVB[i].z - calcVA[i].z * calcVB[i].y;
        viewVolume.face[i].normal.y = calcVA[i].z * calcVB[i].x - calcVA[i].x * calcVB[i].z;
        viewVolume.face[i].normal.z = calcVA[i].x * calcVB[i].y - calcVA[i].y * calcVB[i].x;
    }

    reload = false;

}


void Camera::objRangeCoordTrans(){
    
}
