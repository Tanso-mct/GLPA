#include "hip/hip_runtime.h"
#include "Render.cuh"

Glpa::Render2d::Render2d()
{
}

Glpa::Render2d::~Render2d()
{
}

void Glpa::Render2d::run
(
    std::unordered_map<std::string, Glpa::SceneObject*> objs,
    std::map<int, std::vector<std::string>> drawOrder,
    HDC dc, LPDWORD buf, int bufWidth, int bufHeight, int bufDpi, std::string bgColor
){
    // i = image amount
    // j = this image width x height

    // Separate processing depending on image or text.

    int totalImg = 0;
    int totalText = 0;

    std::vector<int> hImgPosX;
    std::vector<int> hImgPosY;
    std::vector<int> hImgWidth;
    std::vector<int> hImgHeight;
    std::vector<LPDWORD> hImgData;

    int maxImgWidth = 0;
    int maxImgHeight = 0;

    for (auto& pair : drawOrder)
    {
        for (int i = 0; i < pair.second.size(); i++)
        {
            if (Glpa::Image* img = dynamic_cast<Glpa::Image*>(objs[pair.second[i]]))
            {
                if (img->getVisible())
                {
                    Vec2d imgPos = img->getPos();
                    hImgPosX.push_back(imgPos.x);
                    hImgPosY.push_back(imgPos.y);
                    hImgWidth.push_back(img->getWidth());
                    hImgHeight.push_back(img->getHeight());

                    maxImgWidth = (maxImgWidth < img->getWidth()) ? img->getWidth() : maxImgWidth;
                    maxImgHeight = (maxImgHeight < img->getHeight()) ? img->getHeight() : maxImgHeight;

                    DWORD* dData;
                    size_t dataSize = imgPos.x * imgPos.y * sizeof(DWORD);
                    hipMalloc(&dData, dataSize);
                    hipMemcpy(dData, img->getData(), dataSize, hipMemcpyHostToDevice);
                    hImgData.push_back(dData);
                }
            }
        }
    }

    if (hImgData.size() != 0)
    {
        int* dImgPosX;
        int* dImgPosY;
        int* dImgWidth;
        int* dImgHeight;
        LPDWORD* dImgData;
        LPDWORD dBuf;

        DWORD backgroundColor;
        if (bgColor == Glpa::BACKGROUND_BLACK)
        {
            Glpa::Color instColor(255, 255, 255, 1);
            backgroundColor = instColor.GetDword();
        }
        else
        {
            Glpa::Color instColor(255, 255, 255, 1);
            backgroundColor = instColor.GetDword();
        }

        hipMalloc(&dImgPosX, hImgPosX.size() * sizeof(int));
        hipMemcpy(dImgPosX, hImgPosX.data(), hImgPosX.size() * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&dImgPosY, hImgPosY.size() * sizeof(int));
        hipMemcpy(dImgPosY, hImgPosY.data(), hImgPosY.size() * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&dImgWidth, hImgWidth.size() * sizeof(int));
        hipMemcpy(dImgWidth, hImgWidth.data(), hImgWidth.size() * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&dImgHeight, hImgHeight.size() * sizeof(int));
        hipMemcpy(dImgHeight, hImgHeight.data(), hImgHeight.size() * sizeof(int), hipMemcpyHostToDevice);

        hipMalloc(&dImgData, hImgData.size() * sizeof(DWORD*));
        hipMemcpy(dImgData, hImgData.data(), hImgData.size() * sizeof(DWORD*), hipMemcpyHostToDevice);

        hipMalloc(&dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD));
        hipMemcpy(dBuf, buf, bufWidth * bufHeight * bufDpi * sizeof(DWORD), hipMemcpyHostToDevice);

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int dataSizeY = hImgData.size();
        int dataSizeX = maxImgWidth * maxImgHeight;

        int desiredThreadsPerBlockX = 16;
        int desiredThreadsPerBlockY = 16;

        int blocksX = (dataSizeX + desiredThreadsPerBlockX - 1) / desiredThreadsPerBlockX;
        int blocksY = (dataSizeY + desiredThreadsPerBlockY - 1) / desiredThreadsPerBlockY;

        int threadsPerBlockX = min(desiredThreadsPerBlockX, deviceProp.maxThreadsDim[0]);
        int threadsPerBlockY = min(desiredThreadsPerBlockY, deviceProp.maxThreadsDim[1]);

        dim3 dimBlock(threadsPerBlockX, threadsPerBlockY);
        dim3 dimGrid(blocksX, blocksY);

        Gpu2dDraw<<<dimGrid, dimBlock>>>
        (
            dImgPosX, dImgPosY, dImgWidth, dImgHeight, dImgData, hImgData.size(), 
            dBuf, bufWidth, bufHeight, bufDpi, backgroundColor
        );
        hipError_t error = hipGetLastError();
        if (error != 0){
            OutputDebugStringA("GlpaLib ERROR Render.cu - Processing with Cuda failed.\n");
            throw std::runtime_error("Processing with Cuda failed.");
        }

        hipFree(dImgPosX);
        hipFree(dImgPosY);
        hipFree(dImgWidth);
        hipFree(dImgHeight);
        hipFree(dImgData);
        hipFree(dBuf);
    }
    else
    {
        LPDWORD dBuf;

        DWORD backgroundColor;
        if (bgColor == Glpa::BACKGROUND_BLACK)
        {
            Glpa::Color instColor(255, 255, 255, 1);
            backgroundColor = instColor.GetDword();
        }
        else
        {
            Glpa::Color instColor(255, 255, 255, 1);
            backgroundColor = instColor.GetDword();
        }

        hipMalloc(&dBuf, bufWidth * bufHeight * bufDpi * sizeof(DWORD));
        hipMemcpy(dBuf, buf, bufWidth * bufHeight * bufDpi * sizeof(DWORD), hipMemcpyHostToDevice);

        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);

        int dataSizeY = bufWidth;
        int dataSizeX = bufHeight;

        int desiredThreadsPerBlockX = 16;
        int desiredThreadsPerBlockY = 16;

        int blocksX = (dataSizeX + desiredThreadsPerBlockX - 1) / desiredThreadsPerBlockX;
        int blocksY = (dataSizeY + desiredThreadsPerBlockY - 1) / desiredThreadsPerBlockY;

        int threadsPerBlockX = min(desiredThreadsPerBlockX, deviceProp.maxThreadsDim[0]);
        int threadsPerBlockY = min(desiredThreadsPerBlockY, deviceProp.maxThreadsDim[1]);

        dim3 dimBlock(threadsPerBlockX, threadsPerBlockY);
        dim3 dimGrid(blocksX, blocksY);

        Gpu2dDrawBackground<<<dimGrid, dimBlock>>>(dBuf, bufWidth, bufHeight, bufDpi, backgroundColor);
        hipError_t error = hipGetLastError();
        if (error != 0){
            OutputDebugStringA("GlpaLib ERROR Render.cu - Processing with Cuda failed.\n");
            throw std::runtime_error("Processing with Cuda failed.");
        }


        hipFree(dBuf);

    }

}

Glpa::Render3d::Render3d()
{
}

Glpa::Render3d::~Render3d()
{
}

void Glpa::Render3d::run(std::unordered_map<std::string, Glpa::SceneObject*> objs, HDC dc, LPDWORD buf)
{

}

__global__ void Glpa::Gpu2dDraw
(
    int *imgPosX, int *imgPosY, int* imgWidth, int* imgHeight, LPDWORD *imgData, int imgAmount,
    LPDWORD buf, int bufWidth, int bufHeight, int bufDpi, DWORD background
){
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < imgAmount)
    {
        if (j < imgWidth[i] * imgHeight[i])
        {
            /* 
            size = width * height

            point = x + y * width
            
             */

            int drawPoint = imgPosX[i] + imgPosY[i] * bufWidth;
            buf[drawPoint + (j % )];
        }
    }
}

__global__ void Glpa::Gpu2dDrawBackground(LPDWORD buf, int bufWidth, int bufHeight, int bufDpi, DWORD background)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
}
